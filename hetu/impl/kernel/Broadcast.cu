#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void broadcast_kernel(const spec_t* input, size_t input_size,
                                 size_t size, spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    output[idx] = input[idx % input_size];
}

template <typename spec_t>
__global__ void broadcast_gradient_kernel(const spec_t* input,
                                          size_t input_size, size_t size,
                                          spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    output[idx] = input[idx];
}

void BroadcastCuda(const NDArray& input, NDArray& output,
                   const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = output->numel();
  size_t input_size = input->numel();
  if (size == 0 || input_size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "BroadcastCuda", [&]() {
      broadcast_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), input_size, size,
        output->data_ptr<spec_t>());
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

void BroadcastGradientCuda(const NDArray& input, NDArray& output,
                           const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = output->numel();
  size_t input_size = input->numel();
  if (size == 0 || input_size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "BroadcastGradientCuda", [&]() {
      broadcast_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), input_size, size,
        output->data_ptr<spec_t>());
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

} // namespace impl
} // namespace hetu
