#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void concatenate_kernel(const spec_t* input, spec_t* output,
                                   int input_width, int output_width,
                                   int offset, int concat_size, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int post_ind = idx % concat_size;
  int prev_ind = idx / concat_size;
  int mid_ind = prev_ind % input_width + offset;
  prev_ind = prev_ind / input_width;
  int out_ind = (prev_ind * output_width + mid_ind) * concat_size + post_ind;
  output[out_ind] = input[idx];
}

template <typename spec_t>
__global__ void concatenate_gradient_kernel(const spec_t* output_grad,
                                            spec_t* input_grad, int input_width,
                                            int output_width, int offset,
                                            int concat_size, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int post_ind = idx % concat_size;
  int prev_ind = idx / concat_size;
  int mid_ind = prev_ind % input_width + offset;
  prev_ind = prev_ind / input_width;
  int out_ind = (prev_ind * output_width + mid_ind) * concat_size + post_ind;
  input_grad[idx] = output_grad[out_ind];
}

void ConcatenateCuda(const NDArray& input, NDArray& output, size_t axis,
                     size_t offset, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = input->numel();
  int now_ndim = output->ndim();
  HT_ASSERT(input->ndim() == now_ndim);
  int num_concats = 1;
  for (int i = 0; i < axis; ++i) {
    int cur_dim = output->shape(i);
    HT_ASSERT(input->shape(i) == cur_dim);
    num_concats *= cur_dim;
  }
  int concat_size = 1;
  for (int i = axis + 1; i < now_ndim; ++i) {
    int cur_dim = output->shape(i);
    HT_ASSERT(input->shape(i) == cur_dim);
    concat_size *= cur_dim;
  }
  int input_width = input->shape(axis);
  int output_width = output->shape(axis);
  if (size == 0 || input_width == 0 || output_width == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ConcatenateCuda", [&]() {
      concatenate_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), input_width,
        output_width, offset, concat_size, size);
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

void ConcatenateGradientCuda(const NDArray& output_grad, NDArray& input_grad,
                             size_t axis, size_t offset, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(output_grad);
  HT_ASSERT_SAME_DEVICE(output_grad, input_grad);

  size_t size = input_grad->numel();
  int now_ndim = output_grad->ndim();
  HT_ASSERT(now_ndim == input_grad->ndim());
  int num_concats = 1;
  for (int i = 0; i < axis; ++i) {
    int cur_dim = output_grad->shape(i);
    HT_ASSERT(cur_dim == input_grad->shape(i));
    num_concats *= cur_dim;
  }
  int concat_size = 1;
  for (int i = axis + 1; i < now_ndim; ++i) {
    int cur_dim = output_grad->shape(i);
    HT_ASSERT(cur_dim == input_grad->shape(i));
    concat_size *= cur_dim;
  }
  int output_width = output_grad->shape(axis);
  int input_width = input_grad->shape(axis);
  if (size == 0 || input_width == 0 || output_width == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_grad->dtype(), spec_t, "ConcatenateGradientCuda", [&]() {
      concatenate_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        output_grad->data_ptr<spec_t>(), input_grad->data_ptr<spec_t>(),
        input_width, output_width, offset, concat_size, size);
    });
  NDArray::MarkUsedBy({output_grad, input_grad}, stream);
}

} // namespace impl
} // namespace hetu
