#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/stream.h"
#include "hetu/impl/communication/nccl_comm_group.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/stream/CUDAStream.h"

#include <thread>

namespace hetu {
namespace impl {

using namespace hetu::impl::comm;

template <typename spec_t>
__global__ void memory_copy_kernel(const spec_t* input, spec_t* output, size_t size);

void BroadcastCommCuda(const NDArray& input, NDArray& output, int broadcaster,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Sync();
  size_t size = output->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ReshapeCuda", [&]() {
      memory_copy_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), size);
    });
  comm_group->Broadcast(output, broadcaster);
}

void AllReduceCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->AllReduce(input, output);
}

void ReduceCommCuda(const NDArray& input, NDArray& output, int reducer,
                 const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Reduce(input, output, reducer);
}

void AllGatherCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->AllGather(input, output);
}

void ReduceScatterCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->ReduceScatter(input, output);
}

void GatherCuda(const NDArray& input, NDArray& output, int gatherer,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Gather(input, output, gatherer);
}

void ScatterCuda(const NDArray& input, NDArray& output, int scatterer,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Scatter(input, output, scatterer);
}

void P2PSendCuda(const NDArray& data, const Device& dst, const Stream& stream) {
  auto src_rank = GetWorldRank();
  auto dst_rank = DeviceToWorldRank(dst);
  std::vector<int> ranks(2);
  ranks[0] = std::min(src_rank, dst_rank);
  ranks[1] = std::max(src_rank, dst_rank);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Send(data, dst_rank);
}

void P2PRecvCuda(NDArray& data, const Device& src, const Stream& stream) {
  auto src_rank = DeviceToWorldRank(src);
  auto dst_rank = GetWorldRank();
  std::vector<int> ranks(2);
  ranks[0] = std::min(src_rank, dst_rank);
  ranks[1] = std::max(src_rank, dst_rank);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Recv(data, src_rank);
}

} // namespace impl
} // namespace hetu
