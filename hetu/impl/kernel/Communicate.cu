#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/stream.h"
#include "hetu/impl/communication/nccl_comm_group.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/stream/CUDAStream.h"

#include <thread>

namespace hetu {
namespace impl {

using namespace hetu::impl::comm;

template <typename spec_t>
__global__ void memory_copy_kernel(const spec_t* input, spec_t* output, size_t size);

void AllReduceCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->AllReduce(input, output);
}

void AllGatherCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->AllGather(input, output);                  
}

void ReduceScatterCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->ReduceScatter(input, output);
}

void P2PSendCuda(const NDArray& data, const Device& dst, const Stream& stream) {
  auto src_rank = GetWorldRank();
  auto dst_rank = DeviceToWorldRank(dst);
  std::vector<int> ranks(2);
  ranks[0] = std::min(src_rank, dst_rank);
  ranks[1] = std::max(src_rank, dst_rank);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Send(data, dst_rank);
}

void P2PRecvCuda(NDArray& data, const Device& src, const Stream& stream) {
  auto src_rank = DeviceToWorldRank(src);
  auto dst_rank = GetWorldRank();
  std::vector<int> ranks(2);
  ranks[0] = std::min(src_rank, dst_rank);
  ranks[1] = std::max(src_rank, dst_rank);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Recv(data, src_rank);
}

void BatchedISendIRecvCuda(const NDArrayList& send_datas, 
  const std::vector<Device>& dsts, NDArrayList& recv_datas, 
  const std::vector<Device>& srcs, const std::vector<Device>& comm_deivces, 
  const Stream& stream) {
  std::vector<int> ranks(comm_deivces.size());
  std::transform(comm_deivces.begin(), comm_deivces.end(), ranks.begin(), [&](const Device& device) { return DeviceToWorldRank(device); });
  std::sort(ranks.begin(), ranks.end());
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  std::vector<Task> tasks;
  for (int i = 0; i < send_datas.size(); i++) {
    tasks.push_back(comm_group->ISend(send_datas[i], DeviceToWorldRank(dsts[i])));
  }
  for (int i = 0; i < recv_datas.size(); i++) {
    tasks.push_back(comm_group->IRecv(recv_datas[i], DeviceToWorldRank(srcs[i])));
  }
  comm_group->BatchedISendIRecv(tasks);
}

void BroadcastCommCuda(const NDArray& input, NDArray& output, int broadcaster,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Sync();
  size_t size = output->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ReshapeCuda", [&]() {
      memory_copy_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), size);
    });
  comm_group->Broadcast(output, broadcaster);
}

void ReduceCommCuda(const NDArray& input, NDArray& output, int reducer,
                 const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Reduce(input, output, reducer);
}

void GatherCuda(const NDArray& input, NDArray& output, int gatherer,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Gather(input, output, gatherer);
}

void ScatterCuda(const NDArray& input, NDArray& output, int scatterer,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Scatter(input, output, scatterer);
}

} // namespace impl
} // namespace hetu
