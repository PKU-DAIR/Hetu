#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__forceinline__ __device__ void WarpReduceArgmax(spec_t& val) {
  spec_t tmp_val;
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1) {
    tmp_val = __shfl_down_sync(mask, val, k, warpSize);
    if (tmp_val > val) {
      val = tmp_val;
    }
  }
}

template <>
__forceinline__ __device__ void WarpReduceArgmax(bfloat16& val) {
  #if (__CUDA_ARCH__ >= 800)
  bfloat16 tmp_val;
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1) {
    tmp_val = __shfl_down_sync(mask, val, k, warpSize);
    if (tmp_val > val) {
      val = tmp_val;
    }
  }
  #else
  float val_f = float(val);
  float tmp_val;
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1) {
    tmp_val = __shfl_down_sync(mask, val_f, k, warpSize);
    if (tmp_val > val_f) {
      val = bfloat16(tmp_val);
    }
  }
  #endif
}

template <typename spec_t>
__forceinline__ __device__ void BlockReduceArgmax(spec_t& val,
                                                  spec_t* shared_value,
                                                  spec_t* wrap_max,
                                                  size_t idx,
                                                  size_t threads_per_pos) {
  if (threadIdx.x >= idx * threads_per_pos && threadIdx.x < (idx + 1) * threads_per_pos) {

    int thread_id = threadIdx.x % threads_per_pos;
    int wid = thread_id / warpSize;
    int tid = thread_id % warpSize;

    WarpReduceArgmax(val);

    __syncthreads();
    if (tid == 0) {
      shared_value[idx * warpSize + wid] = val;
    }

    __syncthreads();
    val = (thread_id < threads_per_pos / warpSize) ? shared_value[idx * warpSize + tid] : -SIZE_MAX;

    if (wid == 0) {
      WarpReduceArgmax(val);
      if (thread_id == 0)
        wrap_max[idx] = val;
    }
  }
}

template <typename spec_t>
__forceinline__ __device__ spec_t WarpReduceSumExp(spec_t val) {
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1)
    val += __shfl_down_sync(mask, val, k, warpSize);
  return val;
}

template <>
__forceinline__ __device__ bfloat16 WarpReduceSumExp(bfloat16 val) {
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  #if(__CUDA_ARCH__ >= 800)
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1)
    val += __shfl_down_sync(mask, val, k, warpSize);
  #else
  float val_f = float(val);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1)
    val_f += __shfl_down_sync(mask, val_f, k, warpSize);    
  val = bfloat16(val_f);
  #endif
  return val;
}

template <typename spec_t>
__forceinline__ __device__ void BlockReduceSumExp(spec_t& val,
                                                  spec_t* shared,
                                                  spec_t* wrap_sum,
                                                  size_t idx,
                                                  size_t threads_per_pos) {
  if (threadIdx.x >= idx * threads_per_pos && threadIdx.x < (idx + 1) * threads_per_pos) {
    int thread_id = threadIdx.x % threads_per_pos;
    int tid = thread_id % warpSize;
    int wid = thread_id / warpSize;

    val = WarpReduceSumExp(val);

    __syncthreads();
    if (tid == 0)
      shared[idx * threads_per_pos + wid] = val;

    __syncthreads();
    val = (thread_id < threads_per_pos / warpSize) ? shared[idx * threads_per_pos + tid] : 0;

    if (wid == 0) {
      val = WarpReduceSumExp(val);
      if (thread_id == 0)
        wrap_sum[idx] = spec_t(val);
    }
    __syncthreads();
  }
}


template <typename spec_t>
__global__ void softmax_kernel(const spec_t* input, spec_t* output,
                               size_t before_dim_size,
                               size_t reduce_dim_size,
                               size_t after_dim_size) {

  size_t pos_per_block = blockDim.x;
  size_t total_idx = blockIdx.x * pos_per_block + threadIdx.x;
  size_t x = total_idx / after_dim_size;
  size_t y = total_idx % after_dim_size;
  size_t start_ptr, end_ptr, stride;
  size_t pos_tid = 0;
  size_t pos_idx = threadIdx.x;
  if (after_dim_size > 1) {
    stride = after_dim_size;
    start_ptr =
      x * reduce_dim_size * after_dim_size + y + pos_tid * after_dim_size;
    end_ptr = x * reduce_dim_size * after_dim_size + y +
      reduce_dim_size * after_dim_size;
  } else {
    size_t cols_per_thread = reduce_dim_size;
    size_t block_end_ptr = x * reduce_dim_size * after_dim_size + y +
      reduce_dim_size * after_dim_size;
    start_ptr = x * reduce_dim_size * after_dim_size + y +
      pos_tid * cols_per_thread * after_dim_size;
    end_ptr = min(start_ptr + cols_per_thread * after_dim_size, block_end_ptr);
    stride = after_dim_size;
  }
  if (start_ptr >= end_ptr)
    return;

  spec_t max_thread = -SIZE_MAX;
  spec_t sum_thread = 0;
  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    max_thread = hetu::cuda::cuda_max(input[ptr], max_thread);


  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    sum_thread += hetu::cuda::cuda_exp(input[ptr] - max_thread);

  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    output[ptr] = hetu::cuda::cuda_exp(input[ptr] - max_thread) / sum_thread;
}

template <typename spec_t>
__global__ void softmax_kernel2(const spec_t* input, spec_t* output,
                               size_t before_dim_size,
                               size_t reduce_dim_size,
                               size_t after_dim_size,
                               size_t threads_per_pos) {
  __shared__ spec_t shared_sum[1024];
  __shared__ spec_t wrap_max[1024];
  __shared__ spec_t wrap_sum[1024];

  size_t pos_per_block = blockDim.x / threads_per_pos;
  size_t total_idx = blockIdx.x * pos_per_block + threadIdx.x / threads_per_pos;
  size_t x = total_idx / after_dim_size;
  size_t y = total_idx % after_dim_size;
  size_t start_ptr, end_ptr, stride;
  size_t pos_tid = threadIdx.x % threads_per_pos;
  size_t pos_idx = threadIdx.x / threads_per_pos;
  if (after_dim_size > 1) {
    stride = after_dim_size * threads_per_pos;
    start_ptr =
      x * reduce_dim_size * after_dim_size + y + pos_tid * after_dim_size;
    end_ptr = x * reduce_dim_size * after_dim_size + y +
      reduce_dim_size * after_dim_size;
  } else {
    size_t cols_per_thread = (reduce_dim_size + threads_per_pos - 1) / threads_per_pos;
    size_t block_end_ptr = x * reduce_dim_size * after_dim_size + y +
      reduce_dim_size * after_dim_size;
    start_ptr = x * reduce_dim_size * after_dim_size + y +
      pos_tid * cols_per_thread * after_dim_size;
    end_ptr = min(start_ptr + cols_per_thread * after_dim_size, block_end_ptr);
    stride = after_dim_size;
  }
  if (start_ptr >= end_ptr)
    return;

  spec_t max_thread = -SIZE_MAX;
  spec_t sum_thread = 0;
  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    max_thread = hetu::cuda::cuda_max(input[ptr], max_thread);
  
  BlockReduceArgmax(max_thread, shared_sum, wrap_max, pos_idx, threads_per_pos);

  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    sum_thread += hetu::cuda::cuda_exp(input[ptr] - wrap_max[pos_idx]);

  BlockReduceSumExp(sum_thread, shared_sum, wrap_sum, pos_idx, threads_per_pos);
  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    output[ptr] = hetu::cuda::cuda_exp(input[ptr] - wrap_max[pos_idx]) / wrap_sum[pos_idx];
}

void SoftmaxCuda(const NDArray& input, NDArray& output, int64_t dim, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  if (dim < 0) {
    dim = dim + input->ndim();
    HT_ASSERT(dim >= 0 && dim < input->ndim());
  }
  size_t before_dim_size = 1, reduce_dim_size, after_dim_size = 1;
  reduce_dim_size = input->shape(dim);
  for (size_t i = 0; i < input->ndim(); ++i) {
    if (i < dim)
      before_dim_size *= input->shape(i);
    else if (i > dim)
      after_dim_size *= input->shape(i);
  }

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  if (dim != input->ndim() - 1) {
    int blocks = before_dim_size * after_dim_size;
    int threads_per_pos = 1;
    int threads = threads_per_pos;
    while (threads * 2 <= HT_DEFAULT_NUM_THREADS_PER_BLOCK && blocks % 2 == 0) {
      threads *= 2;
      blocks /= 2;
    }
    HT_DISPATCH_FLOATING_TYPES(
      input->dtype(), spec_t, "SoftMaxCuda", [&]() {
        softmax_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          input->data_ptr<spec_t>(), output->data_ptr<spec_t>(),
          before_dim_size, reduce_dim_size, after_dim_size);
      });
  }
  else {
    int blocks = before_dim_size * after_dim_size;
    int threads_per_pos = hetu::impl::GetThreadNum(reduce_dim_size);
    int threads = threads_per_pos;
    while (threads * 2 <= HT_DEFAULT_NUM_THREADS_PER_BLOCK && blocks % 2 == 0) {
      threads *= 2;
      blocks /= 2;
    }
    HT_DISPATCH_FLOATING_TYPES(
      input->dtype(), spec_t, "SoftMaxCuda", [&]() {
        softmax_kernel2<spec_t><<<blocks, threads, 0, hip_stream>>>(
          input->data_ptr<spec_t>(), output->data_ptr<spec_t>(),
          before_dim_size, reduce_dim_size, after_dim_size, threads_per_pos
          );
      }); 
  }
  NDArray::MarkUsedBy({input, output}, stream);
}

template <typename spec_t>
__global__ void softmax_grad_kernel(const spec_t* output, const spec_t* output_grad,
                                    spec_t* input_grad,
                                    size_t before_dim_size,
                                    size_t reduce_dim_size,
                                    size_t after_dim_size) {
  __shared__ spec_t shared_sum[32];
  __shared__ spec_t wrap_sum[1];

  size_t x = blockIdx.x / after_dim_size;
  size_t y = blockIdx.x % after_dim_size;
  size_t start_ptr, end_ptr, stride;
  if (after_dim_size > 1) {
    stride = after_dim_size * blockDim.x;
    start_ptr =
      x * reduce_dim_size * after_dim_size + y + threadIdx.x * after_dim_size;
    end_ptr = x * reduce_dim_size * after_dim_size + y +
      reduce_dim_size * after_dim_size;
  } else {
    size_t cols_per_thread = (reduce_dim_size + blockDim.x - 1) / blockDim.x;
    size_t block_end_ptr = x * reduce_dim_size * after_dim_size + y +
      reduce_dim_size * after_dim_size;
    start_ptr = x * reduce_dim_size * after_dim_size + y +
      threadIdx.x * cols_per_thread * after_dim_size;
    end_ptr = min(start_ptr + cols_per_thread * after_dim_size, block_end_ptr);
    stride = after_dim_size;
  }
  if (start_ptr >= end_ptr)
    return;

  spec_t sum_thread = 0;
  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride)
    sum_thread += output_grad[ptr] * output[ptr];

  BlockReduceSumExp(sum_thread, shared_sum, wrap_sum, 0, blockDim.x);
  for (size_t ptr = start_ptr; ptr < end_ptr; ptr += stride) 
    input_grad[ptr] = output_grad[ptr] * output[ptr] - output[ptr] * wrap_sum[0];
}

void SoftmaxGradientCuda(const NDArray& input_Y, const NDArray& output_grad,
                         NDArray& input_grad, int64_t dim, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_Y);
  HT_ASSERT_SAME_DEVICE(input_Y, output_grad);
  HT_ASSERT_SAME_DEVICE(input_Y, input_grad);

  size_t before_dim_size = 1, reduce_dim_size, after_dim_size = 1;
  reduce_dim_size = input_Y->shape(dim);
  for (size_t i = 0; i < input_Y->ndim(); ++i) {
    if (i < dim)
      before_dim_size *= input_Y->shape(i);
    else if (i > dim)
      after_dim_size *= input_Y->shape(i);
  }

  int blocks = before_dim_size * after_dim_size;
  int threads = hetu::impl::GetThreadNum(reduce_dim_size);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_FLOATING_TYPES(
    input_Y->dtype(), spec_t, "SoftMaxCuda", [&]() {
      softmax_grad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input_Y->data_ptr<spec_t>(), output_grad->data_ptr<spec_t>(),
        input_grad->data_ptr<spec_t>(),
        before_dim_size, reduce_dim_size, after_dim_size);
    });
  NDArray::MarkUsedBy({input_Y, output_grad, input_grad}, stream);
}

} // namespace impl
} // namespace hetu
