#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void binary_cross_entropy_kernel(const spec_t* pred,
                                            const spec_t* label, size_t n_rows,
                                            spec_t* loss) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_rows)
    return;
  spec_t v1 = hetu::cuda::cuda_log(pred[idx]);
  spec_t v2 = hetu::cuda::cuda_log(1 - pred[idx]);
  // clip to -100 following PyTorch
  constexpr spec_t min_value = -100;
  loss[idx] =
    -label[idx] * MAX(v1, min_value) - (1 - label[idx]) * MAX(v2, min_value);
}

template <typename spec_t>
__global__ void
binary_cross_entropy_gradient_kernel(const spec_t* pred, const spec_t* label,
                                     const spec_t* grad_loss, size_t n_rows,
                                     spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_rows)
    return;
  spec_t denominator = pred[idx] * (1 - pred[idx]);
  output[idx] = (pred[idx] - label[idx]) / MAX(denominator, 1e-12);
}

void BinaryCrossEntropyCuda(const NDArray& pred, const NDArray& label,
                            NDArray& loss, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(pred);
  HT_ASSERT_SAME_DEVICE(pred, label);
  HT_ASSERT_SAME_DEVICE(pred, loss);
  HT_ASSERT_SAME_NDIM(pred, label);
  HT_ASSERT_SAME_NDIM(pred, loss);

  size_t n_rows = 1;
  for (size_t i = 0; i < pred->ndim(); i++)
    n_rows *= pred->shape(i);
  if (n_rows == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(n_rows, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(n_rows, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_FLOATING_TYPES(
    pred->dtype(), spec_t, "BinaryCrossEntropyCuda", [&]() {
      binary_cross_entropy_kernel<<<blocks, threads, 0, hip_stream>>>(
        pred->data_ptr<spec_t>(), label->data_ptr<spec_t>(), n_rows,
        loss->data_ptr<spec_t>());
    });
}

void BinaryCrossEntropyGradientCuda(const NDArray& pred, const NDArray& label,
                                    const NDArray& grad_loss, NDArray& output,
                                    const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(pred);
  HT_ASSERT_SAME_DEVICE(pred, label);
  HT_ASSERT_SAME_DEVICE(pred, grad_loss);
  HT_ASSERT_SAME_DEVICE(pred, output);
  HT_ASSERT_SAME_NDIM(pred, label);
  HT_ASSERT_SAME_NDIM(pred, grad_loss);
  HT_ASSERT_SAME_NDIM(pred, output);

  size_t n_rows = 1;
  for (size_t i = 0; i < pred->ndim(); i++)
    n_rows *= pred->shape(i);
  if (n_rows == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(n_rows, 1024);
  blocks.x = DIVUP(n_rows, 1024);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_FLOATING_TYPES(
    pred->dtype(), spec_t, "BinaryCrossEntropyGradientCuda", [&]() {
      binary_cross_entropy_gradient_kernel<<<blocks, threads, 0, hip_stream>>>(
        pred->data_ptr<spec_t>(), label->data_ptr<spec_t>(),
        grad_loss->data_ptr<spec_t>(), n_rows, output->data_ptr<spec_t>());
    });
}

} // namespace impl
} // namespace hetu
