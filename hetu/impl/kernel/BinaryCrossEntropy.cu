#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"
#include "hetu/impl/utils/offset_calculator.cuh"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void binary_cross_entropy_kernel(const spec_t* pred, const spec_t* label,
                                            size_t n_rows, spec_t* loss,
                                            const OffsetCalculator* pred_offset_calculator,
                                            const OffsetCalculator* label_offset_calculator,
                                            const OffsetCalculator* loss_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_rows)
    return;
  auto pred_offset = pred_offset_calculator->get(idx);
  auto label_offset = label_offset_calculator->get(idx);
  auto loss_offset = loss_offset_calculator->get(idx);
  spec_t v1 = hetu::cuda::cuda_log(pred[pred_offset]);
  spec_t v2 = hetu::cuda::cuda_log(1 - pred[pred_offset]);
  // clip to -100 following PyTorch
  spec_t min_value = -100;
  loss[loss_offset] =
    -label[label_offset] * hetu::cuda::cuda_max(v1, min_value) - (1 - label[label_offset]) * hetu::cuda::cuda_max(v2, min_value);
}

template <typename spec_t>
__global__ void
binary_cross_entropy_gradient_kernel(const spec_t* pred, const spec_t* label,
                                     const spec_t* grad_loss, size_t n_rows, spec_t* output,
                                     const OffsetCalculator* pred_offset_calculator,
                                     const OffsetCalculator* label_offset_calculator,
                                     const OffsetCalculator* grad_loss_offset_calculator,
                                     const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_rows)
    return;
  auto pred_offset = pred_offset_calculator->get(idx);
  auto label_offset = label_offset_calculator->get(idx);
  auto grad_loss_offset = grad_loss_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  spec_t denominator = pred[pred_offset] * (1 - pred[pred_offset]);
  output[out_offset] = grad_loss[grad_loss_offset] * (pred[pred_offset] - label[label_offset]) / MAX(denominator, spec_t(1e-12));
}

template <>
__global__ void
binary_cross_entropy_gradient_kernel<bfloat16>(const bfloat16* pred, const bfloat16* label,
                                               const bfloat16* grad_loss, size_t n_rows,
                                               bfloat16* output,
                                               const OffsetCalculator* pred_offset_calculator,
                                               const OffsetCalculator* label_offset_calculator,
                                               const OffsetCalculator* grad_loss_offset_calculator,
                                               const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n_rows)
    return;
  auto pred_offset = pred_offset_calculator->get(idx);
  auto label_offset = label_offset_calculator->get(idx);
  auto grad_loss_offset = grad_loss_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  bfloat16 denominator = pred[pred_offset] * (1 - pred[pred_offset]);
  output[out_offset] = grad_loss[grad_loss_offset] * (pred[pred_offset] - label[label_offset]) / denominator;
}

void BinaryCrossEntropyCuda(const NDArray& pred, const NDArray& label,
                            NDArray& loss, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(pred);
  HT_ASSERT_SAME_DEVICE(pred, label);
  HT_ASSERT_SAME_DEVICE(pred, loss);
  HT_ASSERT_SAME_NDIM(pred, label);
  HT_ASSERT_SAME_NDIM(pred, loss);

  size_t n_rows = 1;
  for (size_t i = 0; i < pred->ndim(); i++)
    n_rows *= pred->shape(i);
  if (n_rows == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(n_rows, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(n_rows, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray pred_offset_calculator_arr, label_offset_calculator_arr,
          loss_offset_calculator_arr;
  OffsetCalculator *pred_offset_calculator, *label_offset_calculator,
                   *loss_offset_calculator;
  std::tie(pred_offset_calculator_arr, pred_offset_calculator) =
    AllocOffsetCalculator(pred, stream);
  std::tie(label_offset_calculator_arr, label_offset_calculator) =
    AllocOffsetCalculator(label, stream);
  std::tie(loss_offset_calculator_arr, loss_offset_calculator) =
    AllocOffsetCalculator(loss, stream);
  HT_DISPATCH_FLOATING_TYPES(
    pred->dtype(), spec_t, "BinaryCrossEntropyCuda", [&]() {
      binary_cross_entropy_kernel<<<blocks, threads, 0, hip_stream>>>(
        pred->data_ptr<spec_t>(), label->data_ptr<spec_t>(), n_rows,
        loss->data_ptr<spec_t>(), pred_offset_calculator,
        label_offset_calculator, loss_offset_calculator);
    });
  NDArray::MarkUsedBy({pred, label, loss, pred_offset_calculator_arr,
                      label_offset_calculator_arr, loss_offset_calculator_arr}, stream);
}

void BinaryCrossEntropyGradientCuda(const NDArray& pred, const NDArray& label,
                                    const NDArray& grad_loss, NDArray& output,
                                    const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(pred);
  HT_ASSERT_SAME_DEVICE(pred, label);
  HT_ASSERT_SAME_DEVICE(pred, grad_loss);
  HT_ASSERT_SAME_DEVICE(pred, output);
  HT_ASSERT_SAME_NDIM(pred, label);
  HT_ASSERT_SAME_NDIM(pred, grad_loss);
  HT_ASSERT_SAME_NDIM(pred, output);

  size_t n_rows = 1;
  for (size_t i = 0; i < pred->ndim(); i++)
    n_rows *= pred->shape(i);
  if (n_rows == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(n_rows, 1024);
  blocks.x = DIVUP(n_rows, 1024);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray pred_offset_calculator_arr, label_offset_calculator_arr,
          grad_loss_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *pred_offset_calculator, *label_offset_calculator,
                   *grad_loss_offset_calculator, *out_offset_calculator;
  std::tie(pred_offset_calculator_arr, pred_offset_calculator) =
    AllocOffsetCalculator(pred, stream);
  std::tie(label_offset_calculator_arr, label_offset_calculator) =
    AllocOffsetCalculator(label, stream);
  std::tie(grad_loss_offset_calculator_arr, grad_loss_offset_calculator) =
    AllocOffsetCalculator(grad_loss, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) =
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_FLOATING_TYPES(
    pred->dtype(), spec_t, "BinaryCrossEntropyGradientCuda", [&]() {
      binary_cross_entropy_gradient_kernel<<<blocks, threads, 0, hip_stream>>>(
        pred->data_ptr<spec_t>(), label->data_ptr<spec_t>(),
        grad_loss->data_ptr<spec_t>(), n_rows, output->data_ptr<spec_t>(),
        pred_offset_calculator, label_offset_calculator,
        grad_loss_offset_calculator, out_offset_calculator);
    });
  NDArray::MarkUsedBy({pred, label, grad_loss, output, pred_offset_calculator_arr,
                      label_offset_calculator_arr, grad_loss_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hetu
