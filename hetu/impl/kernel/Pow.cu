#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"
#include "hetu/impl/utils/offset_calculator.cuh"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void pow_kernel(const spec_t* input, spec_t exponent,
                           size_t size, spec_t* output,
                           const OffsetCalculator* in_offset_calculator,
                           const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto in_offset = in_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  output[out_offset] = hetu::cuda::cuda_pow(input[in_offset], exponent);
}

void PowCuda(const NDArray& input, double exponent, NDArray& output,
             const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, output);

  size_t size = output->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_FLOATING_TYPES(input->dtype(), spec_t, "PowCuda", [&]() {
    pow_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
      input->data_ptr<spec_t>(), static_cast<spec_t>(exponent), size,
      output->data_ptr<spec_t>(), in_offset_calculator,
      out_offset_calculator);
  });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hetu
