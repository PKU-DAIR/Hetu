#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/stream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"
#include "hetu/impl/kernel/Binary.cuh"
#include "hetu/impl/utils/offset_calculator.cuh"

#include <vector>
#include <numeric>

namespace hetu {
namespace impl {

namespace {

inline size_t numel(const HTShape& shape) {
  size_t num = 1;
  for (auto& s : shape) {
    num *= s;
  }
  return num;
}

inline HTStride Shape2Stride(const HTShape& shape) {
  auto size = shape.size();
  HTStride stride(size);
  if (size > 0) {
    stride[size - 1] = 1;
    for (auto d = size - 1; d > 0; d--) {
      stride[d - 1] = stride[d] * shape[d];
    }
  }
  return stride;
}

inline bool maybe_overlapping_memory(const HTShape& shape, const HTStride& stride) {
  if (!shape.empty()) {
    std::vector<size_t> argsort(shape.size());
    std::iota(argsort.begin(), argsort.end(), 0);
    std::sort(
        argsort.begin(), argsort.end(), [&](size_t i, size_t j) {
          return stride[i] < stride[j];
        });

    auto max_index_in_slice = 0;
    for (auto& i : argsort) {
      const auto& stride_i = stride[i];
      if (stride_i <= max_index_in_slice) {
        return true;
      }
      max_index_in_slice += stride_i * (shape[i] - 1);
    }
  }
  return false;
}

} // namespace

// Out-of-place version of as_strided and its gradient
/* It is replaced with in-place version. */
template <typename spec_t>
__global__ void asstrided_kernel(const spec_t* input, spec_t* output,
                                 size_t size, const int64_t* stride_in,
                                 const int64_t* stride_out, int ndim) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int index = 0;
  size_t ind = idx;
  for (int i = 0; i < ndim; i++) {
    int tmp_index = ind / stride_out[i];
    index += tmp_index * stride_in[i];
    ind = ind % stride_out[i];
  }
  output[idx] = input[index];
}

void AsStridedCuda(const NDArray& input, NDArray& output,
                   const HTStride& stride, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  size_t size = output->numel();
  int ndim = output->ndim();
  if (size == 0)
    return;

  auto device_id = input->device().index();
  hetu::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  auto stride_in_arr = hetu::cuda::to_int64_ndarray(stride, device_id);
  auto stride_out_arr =
    hetu::cuda::to_int64_ndarray(output->stride(), device_id);
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "AsStridedCuda", [&]() {
      asstrided_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), size,
        stride_in_arr->data_ptr<int64_t>(), 
        stride_out_arr->data_ptr<int64_t>(), 
        ndim);
    });
  NDArray::MarkUsedBy({input, output, stride_in_arr, stride_out_arr}, stream);
}

template <typename spec_t>
__global__ void asstrided_gradient_kernel(const spec_t* input, spec_t* output,
                                          size_t size, const int64_t* stride_in,
                                          const int64_t* stride_out, int ndim) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int index = 0;
  size_t ind = idx;
  for (int i = 0; i < ndim; i++) {
    int tmp_index = ind / stride_out[i];
    index += tmp_index * stride_in[i];
    ind = ind % stride_out[i];
  }
  hetu::cuda::AtomicAdd(&output[index], input[idx]);
}

void AsStridedGradientCuda(const NDArray& output, NDArray& input,
                           const HTStride& stride, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  size_t size = output->numel();
  int ndim = output->ndim();
  if (size == 0)
    return;

  auto device_id = input->device().index();
  hetu::cuda::CUDADeviceGuard guard(input->device().index());
  CUDAStream hip_stream(stream);
  auto stride_in_arr =
    hetu::cuda::to_int64_ndarray(stride, device_id);
  auto stride_out_arr =
    hetu::cuda::to_int64_ndarray(output->stride(), device_id);
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_FLOATING_TYPES(
    input->dtype(), spec_t, "AsStridedGradientCuda", [&]() {
      spec_t* in_ptr = input->data_ptr<spec_t>();
      CudaMemsetAsync(in_ptr, 0, input->numel() * sizeof(spec_t), hip_stream);
      asstrided_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        output->data_ptr<spec_t>(), in_ptr, size,
        stride_in_arr->data_ptr<int64_t>(), 
        stride_out_arr->data_ptr<int64_t>(), 
        ndim);
    });
  NDArray::MarkUsedBy({input, output, stride_in_arr, stride_out_arr}, stream);
}

// In-place version of as_strided gradient
template <typename spec_a_t, typename spec_b_t, typename Operator>
extern __global__ void binary_elewise_kernel(const spec_a_t* inputA, const spec_b_t* inputB,
                                             size_t size, Operator op, spec_a_t* output,
                                             const OffsetCalculator* A_offset_calculator,
                                             const OffsetCalculator* B_offset_calculator,
                                             const OffsetCalculator* out_offset_calculator);

template <typename spec_t>
__global__ void view_asstrided_gradient_kernel(const spec_t *input, spec_t *output, size_t size,
                                               const int64_t *stride_in, const int64_t *stride_out, 
                                               int ndim, int64_t storage_offset,
                                               const OffsetCalculator* in_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int index = storage_offset;
  size_t ind = idx;
  for (int i = 0; i < ndim; i++) {
    int tmp_index = ind / stride_in[i];
    index += tmp_index * stride_out[i];
    ind -= tmp_index * stride_in[i];
  }
  auto in_offset = in_offset_calculator->get(idx);
  hetu::cuda::AtomicAdd(&output[index], input[in_offset]);
}

template <typename spec_t>
__global__ void count_kernel(spec_t *output, size_t size,
                             const int64_t *stride_in, const int64_t *stride_out,
                             int ndim, int64_t storage_offset) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int index = storage_offset;
  size_t ind = idx;
  spec_t ident = 1;
  for (int i = 0; i < ndim; i++) {
    int tmp_index = ind / stride_in[i];
    index += tmp_index * stride_out[i];
    ind = ind % stride_in[i];
  }
  hetu::cuda::AtomicAdd(&output[index], ident);
}

void AsStridedGradientCuda(const NDArray& grad_output, NDArray& grad_input,
                           const HTShape& out_shape, const HTStride& out_stride,
                           const HTShape& in_shape, const HTStride& in_stride,
                           int64_t in_storage_offset, int64_t out_storage_offset,
                           const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(grad_input);
  HT_ASSERT_SAME_DEVICE(grad_input, grad_output);

  size_t out_size = numel(out_shape);
  if (out_size == 0)
    return;

  auto device_id = grad_input->device().index();
  hetu::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  dim3 blocks, threads;

  size_t odim = out_shape.size();
  HTShape out_stride_contig = Shape2Stride(out_shape);
  auto out_stride_contig_arr = 
    hetu::cuda::to_int64_ndarray(out_stride_contig, device_id);
  auto out_stride_arr =
    hetu::cuda::to_int64_ndarray(out_stride, device_id);
  threads.x = MIN(out_size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(out_size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  NDArray grad_out_offset_calculator_arr;
  OffsetCalculator *grad_out_offset_calculator;
  std::tie(grad_out_offset_calculator_arr, grad_out_offset_calculator) =
    AllocOffsetCalculator(grad_output, stream);
  HT_DISPATCH_FLOATING_TYPES(
    grad_input->dtype(), spec_t, "AsStridedGradientCuda", [&]() {
      view_asstrided_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        grad_output->data_ptr<spec_t>(), grad_input->data_ptr<spec_t>(), out_size,
        out_stride_contig_arr->data_ptr<int64_t>(),
        out_stride_arr->data_ptr<int64_t>(),
        odim, out_storage_offset, grad_out_offset_calculator);
      });

  auto in_maybe_overlap = maybe_overlapping_memory(in_shape, in_stride);
  if (in_maybe_overlap) {
    size_t storage_size = grad_input->storage_size();
    NDArray count = NDArray::zeros({static_cast<int64_t>(storage_size)}, grad_input->device(), grad_input->dtype(),
                                   stream.stream_index());
    
    size_t in_size = numel(in_shape);
    HTShape in_stride_contig = Shape2Stride(in_shape);
    size_t idim = in_shape.size();
    auto in_stride_contig_arr =
      hetu::cuda::to_int64_ndarray(in_stride_contig, device_id);
    auto in_stride_arr = 
      hetu::cuda::to_int64_ndarray(in_stride, device_id);
    threads.x = MIN(in_size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
    blocks.x = DIVUP(in_size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
    HT_DISPATCH_FLOATING_TYPES(
      grad_input->dtype(), spec_t, "CountCuda", [&]() {
        count_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          count->data_ptr<spec_t>(), in_size,
          in_stride_contig_arr->data_ptr<int64_t>(),
          in_stride_arr->data_ptr<int64_t>(),
          idim, in_storage_offset);
        });
    NDArray grad_in_offset_calculator_arr, cnt_offset_calculator_arr;
    OffsetCalculator *grad_in_offset_calculator, *cnt_offset_calculator;
    std::tie(grad_in_offset_calculator_arr, grad_in_offset_calculator) =
      AllocOffsetCalculator(grad_input, stream);
    std::tie(cnt_offset_calculator_arr, cnt_offset_calculator) =
      AllocOffsetCalculator(count, stream);
    threads.x = MIN(storage_size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
    blocks.x = DIVUP(storage_size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
    HT_DISPATCH_FLOATING_TYPES(
      grad_input->dtype(), spec_t, "BinaryElewiseCuda", [&]() {
        binary_elewise_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          grad_input->data_ptr<spec_t>(), count->data_ptr<spec_t>(), storage_size,
          kdivides<spec_t, spec_t>(), grad_input->data_ptr<spec_t>(),
          grad_in_offset_calculator, cnt_offset_calculator, grad_in_offset_calculator);
        });

    NDArray::MarkUsedBy({count, in_stride_contig_arr, in_stride_arr,
                        grad_in_offset_calculator_arr, cnt_offset_calculator_arr}, stream);
  }

  auto output_meta = NDArrayMeta().set_dtype(grad_input->dtype())
                                  .set_shape(in_shape)
                                  .set_stride(in_stride)
                                  .set_device(grad_input->device());
  grad_input = NDArray(output_meta, grad_input->storage(), in_storage_offset);

  NDArray::MarkUsedBy({grad_input, grad_output, out_stride_contig_arr,
                      out_stride_arr, grad_out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hetu
