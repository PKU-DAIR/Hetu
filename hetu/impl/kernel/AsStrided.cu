#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void asstrided_kernel(const spec_t *input, spec_t *output, size_t size,
                                 int64_t *stride_in, int64_t *stride_out, int ndim) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int index = 0;
  size_t ind = idx;
  for (int i = 0; i < ndim; i++) {
    int tmp_index = ind / stride_out[i];
    index += tmp_index * stride_in[i];
    ind = ind % stride_out[i];
  }
  output[idx] = input[index];
}


void AsStridedCuda(const NDArray& input, NDArray& output, HTShape stride, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = output->numel();
  int ndim = output->ndim();
  int64_t *stride_in = NULL;
  int64_t *stride_out = NULL;
  size_t buf_size = 3 * ndim * sizeof(int64_t);
  DataPtr stride_in_ptr = AllocFromMemoryPool(input->device(), buf_size);
  stride_in = (int64_t*) stride_in_ptr.ptr;
  DataPtr stride_out_ptr = AllocFromMemoryPool(input->device(), buf_size);
  stride_out = (int64_t*) stride_out_ptr.ptr;
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  CudaMemcpyAsync(stride_in, (void*) stride.data(), buf_size, hipMemcpyHostToDevice, hip_stream);
  CudaMemcpyAsync(stride_out, (void*) output->stride().data(), buf_size, hipMemcpyHostToDevice, hip_stream);
  HT_DISPATCH_FLOATING_TYPES(
    input->dtype(), spec_t, "AsStridedCuda", [&]() {
      asstrided_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), size, stride_in, stride_out, ndim);
    });
  FreeToMemoryPool(stride_in_ptr);
  FreeToMemoryPool(stride_out_ptr);
}

template <typename spec_t>
extern __global__ void array_zero_set_kernel(spec_t* input, size_t size);

template <typename spec_t>
__global__ void asstrided_gradient_kernel(const spec_t *input, spec_t *output, size_t size,
                                          int64_t *stride_in, int64_t *stride_out, int ndim) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int index = 0;
  size_t ind = idx;
  for (int i = 0; i < ndim; i++) {
    int tmp_index = ind / stride_out[i];
    index += tmp_index * stride_in[i];
    ind = ind % stride_out[i];
  }
  hetu::cuda::AtomicAdd(&output[index], input[idx]);
}

void AsStridedGradientCuda(const NDArray& output, NDArray& input, HTShape stride, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = output->numel();
  int ndim = output->ndim();
  int64_t *stride_in = NULL;
  int64_t *stride_out = NULL;
  size_t buf_size = 3 * ndim * sizeof(int64_t);
  DataPtr stride_in_ptr = AllocFromMemoryPool(input->device(), buf_size);
  stride_in = (int64_t*) stride_in_ptr.ptr;
  DataPtr stride_out_ptr = AllocFromMemoryPool(input->device(), buf_size);
  stride_out = (int64_t*) stride_out_ptr.ptr;
  if (size == 0)
    return;
  dim3 blocks, threads;
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  threads.x = MIN(input->numel(), HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(input->numel(), HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_FLOATING_TYPES(
    output->dtype(), spec_t, "ArraySetZeroCuda", [&]() {
      array_zero_set_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), input->numel());
    });
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CudaMemcpyAsync(stride_in, (void*) stride.data(), buf_size, hipMemcpyHostToDevice, hip_stream);
  CudaMemcpyAsync(stride_out, (void*) output->stride().data(), buf_size, hipMemcpyHostToDevice, hip_stream);
  HT_DISPATCH_FLOATING_TYPES(
    input->dtype(), spec_t, "AsStridedGradientCuda", [&]() {
      asstrided_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        output->data_ptr<spec_t>(), input->data_ptr<spec_t>(), size, stride_in, stride_out, ndim);
    });
  FreeToMemoryPool(stride_in_ptr);
  FreeToMemoryPool(stride_out_ptr);
}


} // namespace impl
} // namespace hetu
