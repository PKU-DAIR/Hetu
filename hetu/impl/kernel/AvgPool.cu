#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

void AvgPoolCuda(const NDArray& input, const size_t kernel_H,
                 const size_t kernel_W, NDArray& output, const size_t padding,
                 const size_t stride, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_CUDA_DEVICE(output);
  HT_ASSERT_SAME_DEVICE(input, output);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());
  // input
  size_t input_N = input->shape(0);
  size_t input_C = input->shape(1);
  size_t input_H = input->shape(2);
  size_t input_W = input->shape(3);

  // output
  size_t output_H = output->shape(2);
  size_t output_W = output->shape(3);

  hipdnnDataType_t datatype;
  if (input->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (input->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "AvgPoolCuda", [&]() {
      const spec_t* input_data = (const spec_t*) input->data_ptr<spec_t>();
      spec_t* output_data = (spec_t*) output->data_ptr<spec_t>();
      // pooling descriptor
      hipdnnPoolingDescriptor_t avgpool_desc;
      CUDNN_CALL(hipdnnCreatePoolingDescriptor(&avgpool_desc));
      CUDNN_CALL(hipdnnSetPooling2dDescriptor(
        avgpool_desc, HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
        HIPDNN_PROPAGATE_NAN, kernel_H, kernel_W, padding, padding, stride,
        stride));

      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));

      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C,
                                            output_H, output_W));

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      CUDNN_CALL(hipdnnPoolingForward(handle, avgpool_desc, &alpha, input_desc,
                                     input_data, &beta, output_desc,
                                     output_data));

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyPoolingDescriptor(avgpool_desc));
    });
}

void AvgPoolGradientCuda(const NDArray& output_Y, const NDArray& gradient_Y,
                         const NDArray& input_X, const size_t kernel_H,
                         const size_t kernel_W, NDArray& gradient_X,
                         const size_t padding, const size_t stride,
                         const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(output_Y);
  HT_ASSERT_CUDA_DEVICE(gradient_Y);
  HT_ASSERT_CUDA_DEVICE(input_X);
  HT_ASSERT_CUDA_DEVICE(gradient_X);
  HT_ASSERT_SAME_DEVICE(output_Y, gradient_Y);
  HT_ASSERT_SAME_DEVICE(output_Y, input_X);
  HT_ASSERT_SAME_DEVICE(output_Y, gradient_X);

  CUDAStream hip_stream(stream);
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  // input
  size_t input_N = input_X->shape(0);
  size_t input_C = input_X->shape(1);
  size_t input_H = input_X->shape(2);
  size_t input_W = input_X->shape(3);
  // output
  size_t output_H = output_Y->shape(2);
  size_t output_W = output_Y->shape(3);

  hipdnnDataType_t datatype;
  if (output_Y->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (output_Y->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  } else {
    HT_VALUE_ERROR << "Invalid Datatype.";
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    output_Y->dtype(), spec_t, "AvgPoolGradientCuda", [&]() {
      const spec_t* input_data = (const spec_t*) input_X->data_ptr<spec_t>();
      spec_t* gradient_x_data = (spec_t*) gradient_X->data_ptr<spec_t>();
      const spec_t* output_data = (const spec_t*) output_Y->data_ptr<spec_t>();
      const spec_t* gradient_Y_data =
        (const spec_t*) gradient_Y->data_ptr<spec_t>();
      // pooling descriptor
      hipdnnPoolingDescriptor_t avgpool_desc;
      CUDNN_CALL(hipdnnCreatePoolingDescriptor(&avgpool_desc));
      CUDNN_CALL(hipdnnSetPooling2dDescriptor(
        avgpool_desc, HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
        HIPDNN_PROPAGATE_NAN, kernel_H, kernel_W, padding, padding, stride,
        stride));

      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));

      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C,
                                            output_H, output_W));

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      CUDNN_CALL(hipdnnPoolingBackward(handle, avgpool_desc, &alpha, output_desc,
                                      output_data, output_desc, gradient_Y_data,
                                      input_desc, input_data, &beta, input_desc,
                                      gradient_x_data));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyPoolingDescriptor(avgpool_desc));
    });
}

} // namespace impl
} // namespace hetu
