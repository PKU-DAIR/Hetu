#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void memory_copy_kernel(const spec_t* input, spec_t* output,
                                   size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = input[idx];
}

void ReshapeCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t input_size = input->numel();
  size_t size = output->numel();
  HT_ASSERT(input_size == size) << "input size and output size are different. "
                                << input_size << " " << size;
  if (input_size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ReshapeCuda", [&]() {
      memory_copy_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), size);
    });
}

} // namespace impl
} // namespace hetu
