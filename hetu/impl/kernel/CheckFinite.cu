#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
extern  __global__ void array_set_kernel(spec_t* arr, spec_t value, size_t size);

template <typename spec_t>
__global__ void check_finite_kernel(const spec_t* input, size_t size, float* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  if (!isfinite(float(input[idx])))
    output[0] = 1.f;
}

void CheckFiniteCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  // HT_LOG_INFO << input << "\n" << output;
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "CheckFiniteCuda", [&]() {
      array_set_kernel<float><<<blocks, threads, 0, hip_stream>>>(
        output->data_ptr<float>(), 0, 1);
      check_finite_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), size, output->data_ptr<float>());
    });
  // HT_LOG_INFO << input << "\n" << output;
}

} // namespace impl
} // namespace hetu
