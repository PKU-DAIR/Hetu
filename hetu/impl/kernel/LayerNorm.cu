#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/kernel/Binary.cuh"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_t, typename Operator>
extern __global__ void binary_elewise_kernel(const spec_t* inputA, const spec_t* inputB,
                                             size_t size, Operator op, spec_t* output);

template <typename spec_t>
__forceinline__ __device__ spec_t WarpReduceSum(spec_t val) {
  unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
  for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1)
    val += __shfl_down_sync(mask, val, k, warpSize);
  return val;
}

template <typename spec_t>
__forceinline__ __device__ void BlockReduceSum(spec_t& val, spec_t* shared) {
  int tid = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = WarpReduceSum(val);

  __syncthreads();
  if (tid == 0)
    shared[wid] = val;

  __syncthreads();
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[tid] : 0;

  if (wid == 0)
    val = WarpReduceSum(val);
}

template <typename spec_t>
__global__ void layer_norm_kernel(const spec_t* x, const spec_t* scale,
                                  const spec_t* bias, spec_t* y, spec_t* mean,
                                  spec_t* var, const float eps,
                                  const int last_dim) {
  __shared__ spec_t var_share;
  __shared__ spec_t mean_share;
  __shared__ spec_t shared_var[32];
  __shared__ spec_t shared_mean[32];

  int begin = blockIdx.x * last_dim + threadIdx.x;
  int end = (blockIdx.x + 1) * last_dim;

  spec_t mean_thread = 0, var_thread = 0;
  for (int i = begin; i < end; i += blockDim.x) {
    mean_thread += x[i];
    var_thread += (x[i] * x[i]);
  }

  BlockReduceSum(mean_thread, shared_mean);
  BlockReduceSum(var_thread, shared_var);
  if (threadIdx.x == 0) {
    mean[blockIdx.x] = mean_share = mean_thread / last_dim;
    var_share = var_thread / last_dim - mean_share * mean_share;
    if (double(var_share) < 0)
      var_share = 0;
    var[blockIdx.x] = var_share;
  }
  __syncthreads();

  mean_thread = mean_share;
  var_thread = var_share;
  spec_t tmp = 1.0f / sqrtf(var_thread + eps);
  for (int i = begin, j = threadIdx.x; i < end;
       i += blockDim.x, j += blockDim.x)
    y[i] = (x[i] - mean_thread) * tmp * scale[j] + bias[j];
}

void LayerNormCuda(const NDArray& in_arr, const NDArray& ln_scale,
                   const NDArray& ln_bias, NDArray& mean_arr, NDArray& var_arr,
                   NDArray& out_arr, int64_t reduce_dims, 
                   float eps, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(in_arr);
  HT_ASSERT_SAME_DEVICE(in_arr, ln_scale);
  HT_ASSERT_SAME_DEVICE(in_arr, ln_bias);
  HT_ASSERT_SAME_DEVICE(in_arr, mean_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, var_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, out_arr);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  int ndim = in_arr->ndim();
  int base_dim = 1, last_dim = 1;
  for (int i = 0; i < ndim - reduce_dims; ++i)
    base_dim *= in_arr->shape(i);
  for (int i = ndim - reduce_dims; i < ndim; ++i)
    last_dim *= in_arr->shape(i);
  // int BlockDim = (last_dim >= 1024 ? 1024: 64);
  dim3 blocks, threads;
  threads.x = (last_dim >= 1024 ? 1024 : 64);
  blocks.x = base_dim;
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "LayerNormCuda", [&]() {
      layer_norm_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        in_arr->data_ptr<spec_t>(), ln_scale->data_ptr<spec_t>(),
        ln_bias->data_ptr<spec_t>(), out_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(), eps,
        last_dim);
    });
  return;
}

template <typename spec_t>
__global__ void calculate_gscale(const spec_t* grads, const spec_t* in_arr,
                                 const spec_t* mean_arr, const spec_t* var_arr,
                                 spec_t* grad_scale, spec_t eps,
                                 int last_dim, size_t size) {
  size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind >= size)
    return;
  int mo_ind = ind / last_dim;
  spec_t std = sqrtf(var_arr[mo_ind] + eps);
  spec_t x_centered = in_arr[ind] - mean_arr[mo_ind];
  spec_t x_norm = x_centered / std;
  grad_scale[ind] = grads[ind] * x_norm;
}

template <typename spec_t>
__global__ void calculate_grad_kernel(const spec_t* out_grads,
                                      const spec_t* in_arr,
                                      const spec_t* mean_arr,
                                      const spec_t* var_arr, 
                                      spec_t* ds, spec_t* db,
                                      spec_t* grad_arr,
                                      size_t lastdim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / lastdim;
  // float y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  spec_t tmp = (db[mo_idx] * mean_arr[mo_idx] - ds[mo_idx]) * (in_arr[idx] - mean_arr[mo_idx]) /
                (var_arr[mo_idx] + eps);
  grad_arr[idx] = (out_grads[idx] + (tmp - db[mo_idx]) / (spec_t)lastdim) / 
    hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

void LayerNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
                           const NDArray& ln_scale, NDArray& grad_arr,
                           NDArray& grad_scale, NDArray& grad_bias,
                           const NDArray& mean_arr, const NDArray& var_arr,
                           int64_t reduce_dims, float eps, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(out_grads);
  HT_ASSERT_SAME_DEVICE(out_grads, ln_scale);
  HT_ASSERT_SAME_DEVICE(out_grads, in_arr);
  HT_ASSERT_SAME_DEVICE(out_grads, mean_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, var_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, grad_scale);
  HT_ASSERT_SAME_DEVICE(out_grads, grad_arr);
  HT_ASSERT_SAME_DEVICE(out_grads, grad_bias);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  int ndim = out_grads->ndim();
//   HT_ASSERT(ndim == 4);
  size_t total_elements = 1;

  hipdnnDataType_t datatype;
  hipdnnIndicesType_t indicetype;
  if (in_arr->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
    indicetype = HIPDNN_32BIT_INDICES;
  } else if (in_arr->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
    indicetype = HIPDNN_64BIT_INDICES;
  }

  int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);
  size_t cpu_mem = ndim * sizeof(int);
  int* dimA = (int*) malloc(cpu_mem);
  int* strideA = (int*) malloc(cpu_mem);
  int* dimB = (int*) malloc(cpu_mem);
  int* strideB = (int*) malloc(cpu_mem);
  int* dimC = (int*) malloc(cpu_mem);
  int* strideC = (int*) malloc(cpu_mem);

  int temp_strideA = 1;
  int temp_strideB = 1;
  int temp_strideC = 1;

  for (int i = ndim - 1; i >= 0; --i) {
    dimA[i] = (int) in_arr->shape(i);
    dimB[i] = i >= in_arr->ndim() - reduce_dims ? (int) in_arr->shape(i) : 1;
    dimC[i] = i < in_arr->ndim() - reduce_dims ? (int) in_arr->shape(i) : 1;
    strideA[i] = temp_strideA;
    strideB[i] = temp_strideB;
    strideC[i] = temp_strideC;
    temp_strideA *= dimA[i];
    temp_strideB *= dimB[i];
    temp_strideC *= dimC[i];
  }

  for (int i = 0; i < ndim; ++i)
    total_elements *= out_grads->shape(i);
  int lastdim = 1;
  for (size_t i = 0; i < reduce_dims; ++i) {
    lastdim *= out_grads->shape(ndim - 1 -i);
  }

  size_t size = total_elements;
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "CauculateGradCuda", [&]() {
      spec_t* ds = NULL;
      DataPtr ds_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideC * sizeof(spec_t));
      ds = (spec_t*) ds_ptr.ptr;

      spec_t* db = NULL;
      DataPtr db_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideC * sizeof(spec_t));
      db = (spec_t*) db_ptr.ptr;

      spec_t* dy_mul_x = NULL;
      DataPtr dy_mul_x_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideA * sizeof(spec_t));
      dy_mul_x = (spec_t*) dy_mul_x_ptr.ptr;

      DataPtr gscale_ptr = AllocFromMemoryPool(out_grads->device(), temp_strideA * sizeof(spec_t));
      spec_t* gscale = (spec_t*) gscale_ptr.ptr;

      DataPtr workspace_ptr = AllocFromMemoryPool(out_grads->device(), temp_strideA * sizeof(spec_t));
      spec_t* workspace = (spec_t*) workspace_ptr.ptr;

      float one = 1.0f;
      float zero = 0.0f;

      hipdnnReduceTensorDescriptor_t rtd;
      CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
      CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
        rtd, HIPDNN_REDUCE_TENSOR_ADD, datatype, HIPDNN_PROPAGATE_NAN,
        HIPDNN_REDUCE_TENSOR_NO_INDICES, indicetype));

      hipdnnTensorDescriptor_t adesc;
      hipdnnTensorDescriptor_t bdesc;
      hipdnnTensorDescriptor_t cdesc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&adesc));
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&bdesc));
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&cdesc));

      CUDNN_CALL(
        hipdnnSetTensorNdDescriptor(adesc, datatype, ndim, dimA, strideA));
      CUDNN_CALL(
        hipdnnSetTensorNdDescriptor(bdesc, datatype, ndim, dimB, strideB));
      CUDNN_CALL(
        hipdnnSetTensorNdDescriptor(cdesc, datatype, ndim, dimC, strideC));
      
      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
        adesc, (const void*) out_grads->data_ptr<void>(), &zero, bdesc,
        (void*) grad_bias->data_ptr<void>()));

      calculate_gscale<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
        gscale, eps, lastdim, (size_t) temp_strideA);

      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
        adesc, (const void*) gscale, &zero, bdesc,
        (void*) grad_scale->data_ptr<void>())); 
      
      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
        adesc, (const void*) out_grads->data_ptr<void>(), &zero, cdesc,
        (void*) db));      

      auto op = kmultiplies<spec_t>();

      binary_elewise_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        size, op, dy_mul_x);
      
      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
        adesc, (const void*) dy_mul_x, &zero, cdesc,
        (void*) ds));  
        

      calculate_grad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
        ds, db,
        grad_arr->data_ptr<spec_t>(), lastdim, eps, size);
      FreeToMemoryPool(ds_ptr);
      FreeToMemoryPool(db_ptr);
      FreeToMemoryPool(dy_mul_x_ptr);
      FreeToMemoryPool(gscale_ptr);
      FreeToMemoryPool(workspace_ptr);
    });
    free(dimA);
    free(strideA);
    free(dimB);
    free(strideB);
    free(dimC);
    free(strideC);
}

} // namespace impl
} // namespace hetu
