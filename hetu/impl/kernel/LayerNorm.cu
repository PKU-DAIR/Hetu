#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/kernel/Binary.cuh"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_a_t, typename spec_b_t, typename Operator>
extern __global__ void binary_elewise_kernel(const spec_a_t* inputA, const spec_b_t* inputB,
                                             size_t size, Operator op, spec_a_t* output);

// template <typename spec_t>
// __forceinline__ __device__ spec_t WarpReduceSum(spec_t val) {
//   unsigned int mask = __ballot_sync(0xFFFFFFFF, true);
//   for (unsigned int k = (warpSize >> 1); k > 0; k >>= 1)
//     val += __shfl_down_sync(mask, val, k, warpSize);
//   return val;
// }

// template <typename spec_t>
// __forceinline__ __device__ void BlockReduceSum(spec_t& val, spec_t* shared) {
//   int tid = threadIdx.x % warpSize;
//   int wid = threadIdx.x / warpSize;

//   val = WarpReduceSum(val);

//   __syncthreads();
//   if (tid == 0)
//     shared[wid] = val;

//   __syncthreads();
//   val = (threadIdx.x < blockDim.x / warpSize) ? spec_t(shared[tid]) : spec_t(0);

//   if (wid == 0)
//     val = WarpReduceSum(val);
// }

template <typename spec_t>
__global__ void layer_norm_kernel(const spec_t* x, const spec_t* scale,
                                  const spec_t* bias, spec_t* y, spec_t* mean,
                                  spec_t* var, const float eps,
                                  const int last_dim) {
  __shared__ spec_t var_share;
  __shared__ spec_t mean_share;
  __shared__ spec_t shared_var[32];
  __shared__ spec_t shared_mean[32];

  int begin = blockIdx.x * last_dim + threadIdx.x;
  int end = (blockIdx.x + 1) * last_dim;

  spec_t mean_thread = 0, var_thread = 0;
  for (int i = begin; i < end; i += blockDim.x) {
    mean_thread += x[i];
    var_thread += (x[i] * x[i]);
  }

  hetu::cuda::BlockReduceSum(mean_thread, shared_mean);
  hetu::cuda::BlockReduceSum(var_thread, shared_var);
  if (threadIdx.x == 0) {
    mean[blockIdx.x] = mean_share = mean_thread / last_dim;
    var_share = var_thread / last_dim - mean_share * mean_share;
    if (double(var_share) < 0)
      var_share = 0;
    var[blockIdx.x] = var_share;
  }
  __syncthreads();

  mean_thread = mean_share;
  var_thread = var_share;
  spec_t tmp = 1.0f / sqrtf(var_thread + eps);
  for (int i = begin, j = threadIdx.x; i < end;
       i += blockDim.x, j += blockDim.x)
    y[i] = (x[i] - mean_thread) * tmp * scale[j] + bias[j];
}

void LayerNormCuda(const NDArray& in_arr, const NDArray& ln_scale,
                   const NDArray& ln_bias, NDArray& mean_arr, NDArray& var_arr,
                   NDArray& out_arr, int64_t reduce_dims, 
                   float eps, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(in_arr);
  HT_ASSERT_SAME_DEVICE(in_arr, ln_scale);
  HT_ASSERT_SAME_DEVICE(in_arr, ln_bias);
  HT_ASSERT_SAME_DEVICE(in_arr, mean_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, var_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, out_arr);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  int ndim = in_arr->ndim();
  int base_dim = 1, last_dim = 1;
  for (int i = 0; i < ndim - reduce_dims; ++i)
    base_dim *= in_arr->shape(i);
  for (int i = ndim - reduce_dims; i < ndim; ++i)
    last_dim *= in_arr->shape(i);
  // int BlockDim = (last_dim >= 1024 ? 1024: 64);
  dim3 blocks, threads;
  threads.x = (last_dim >= 1024 ? 1024 : 64);
  blocks.x = base_dim;
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "LayerNormCuda", [&]() {
      layer_norm_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        in_arr->data_ptr<spec_t>(), ln_scale->data_ptr<spec_t>(),
        ln_bias->data_ptr<spec_t>(), out_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(), eps,
        last_dim);
    });
  return;
}

template <typename spec_t>
__global__ void calculate_gscale(const spec_t* grads, const spec_t* in_arr,
                                 const spec_t* mean_arr, const spec_t* var_arr,
                                 spec_t* grad_scale, float eps,
                                 int last_dim, size_t size) {
  size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind >= size)
    return;
  int mo_ind = ind / last_dim;
  spec_t std = hetu::cuda::cuda_sqrt(var_arr[mo_ind] + eps);
  spec_t x_centered = in_arr[ind] - mean_arr[mo_ind];
  spec_t x_norm = x_centered / std;
  grad_scale[ind] = grads[ind] * x_norm;
}

template <>
__global__ void calculate_gscale<float16>(const float16* grads, const float16* in_arr,
                                          const float16* mean_arr, const float16* var_arr,
                                          float16* grad_scale, float eps,
                                          int last_dim, size_t size) {
  size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind >= size)
    return;
  int mo_ind = ind / last_dim;
  float16 std = hetu::cuda::cuda_sqrt(var_arr[mo_ind] + eps);
  float16 x_centered = in_arr[ind] - mean_arr[mo_ind];
  float16 x_norm = x_centered / std;
  grad_scale[ind] = grads[ind] * x_norm;
}

template <typename spec_t>
__global__ void calculate_grad_kernel_layer(const spec_t* out_grads,
                                      const spec_t* in_arr,
                                      const spec_t* mean_arr,
                                      const spec_t* var_arr, 
                                      spec_t* ds, spec_t* db,
                                      spec_t* grad_arr,
                                      size_t lastdim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / lastdim;
  // float y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  spec_t tmp = (db[mo_idx] * mean_arr[mo_idx] - ds[mo_idx]) * (in_arr[idx] - mean_arr[mo_idx]) /
                (var_arr[mo_idx] + eps);
  grad_arr[idx] = (out_grads[idx] + (tmp - db[mo_idx]) / (spec_t)lastdim) / 
    hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

template <>
__global__ void calculate_grad_kernel_layer<float16>(const float16* out_grads,
                                      const float16* in_arr,
                                      const float16* mean_arr,
                                      const float16* var_arr, 
                                      float16* ds, float16* db,
                                      float16* grad_arr,
                                      size_t lastdim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / lastdim;
  // float y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  float16 tmp = (db[mo_idx] * mean_arr[mo_idx] - ds[mo_idx]) * (in_arr[idx] - mean_arr[mo_idx]) /
                (var_arr[mo_idx] + eps);
  grad_arr[idx] = (out_grads[idx] + (tmp - db[mo_idx]) / (float16)lastdim) / 
    hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

// void LayerNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
//                            const NDArray& ln_scale, NDArray& grad_arr,
//                            NDArray& grad_scale, NDArray& grad_bias,
//                            const NDArray& mean_arr, const NDArray& var_arr,
//                            int64_t reduce_dims, float eps, const Stream& stream) {
//   HT_ASSERT_CUDA_DEVICE(out_grads);
//   HT_ASSERT_SAME_DEVICE(out_grads, ln_scale);
//   HT_ASSERT_SAME_DEVICE(out_grads, in_arr);
//   HT_ASSERT_SAME_DEVICE(out_grads, mean_arr); 
//   HT_ASSERT_SAME_DEVICE(out_grads, var_arr); 
//   HT_ASSERT_SAME_DEVICE(out_grads, grad_scale);
//   HT_ASSERT_SAME_DEVICE(out_grads, grad_arr);
//   HT_ASSERT_SAME_DEVICE(out_grads, grad_bias);

//   CUDAStream hip_stream(stream);
//   hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
//   hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

//   int ndim = out_grads->ndim();
// //   HT_ASSERT(ndim == 4);
//   size_t total_elements = 1;

//   hipdnnDataType_t datatype;
//   hipdnnIndicesType_t indicetype;
//   if (in_arr->dtype() == DataType::FLOAT32) {
//     datatype = HIPDNN_DATA_FLOAT;
//     indicetype = HIPDNN_32BIT_INDICES;
//   } else if (in_arr->dtype() == DataType::FLOAT64) {
//     datatype = HIPDNN_DATA_DOUBLE;
//     indicetype = HIPDNN_64BIT_INDICES;
//   } else if (in_arr->dtype() == DataType::FLOAT16) {
//     datatype = HIPDNN_DATA_HALF;
//     indicetype = HIPDNN_32BIT_INDICES;
//   }
//   #if defined(CUDNN_VERSION) && CUDNN_VERSION >= 8200
//   else if (in_arr->dtype() == DataType::BFLOAT16) {
//     datatype = CUDNN_DATA_BFLOAT16;
//     indicetype = HIPDNN_32BIT_INDICES;
//   }
//   #endif
//   else {
//     HT_LOG_INFO << "UNSUPPORTED TYPE:" << in_arr->dtype();
//   }

//   int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);
//   size_t cpu_mem = ndim * sizeof(int);
//   int* dimA = (int*) malloc(cpu_mem);
//   int* strideA = (int*) malloc(cpu_mem);
//   int* dimB = (int*) malloc(cpu_mem);
//   int* strideB = (int*) malloc(cpu_mem);
//   int* dimC = (int*) malloc(cpu_mem);
//   int* strideC = (int*) malloc(cpu_mem);

//   int temp_strideA = 1;
//   int temp_strideB = 1;
//   int temp_strideC = 1;

//   for (int i = ndim - 1; i >= 0; --i) {
//     dimA[i] = (int) in_arr->shape(i);
//     dimB[i] = i >= in_arr->ndim() - reduce_dims ? (int) in_arr->shape(i) : 1;
//     dimC[i] = i < in_arr->ndim() - reduce_dims ? (int) in_arr->shape(i) : 1;
//     strideA[i] = temp_strideA;
//     strideB[i] = temp_strideB;
//     strideC[i] = temp_strideC;
//     temp_strideA *= dimA[i];
//     temp_strideB *= dimB[i];
//     temp_strideC *= dimC[i];
//   }

//   for (int i = 0; i < ndim; ++i)
//     total_elements *= out_grads->shape(i);
//   int lastdim = 1;
//   for (size_t i = 0; i < reduce_dims; ++i) {
//     lastdim *= out_grads->shape(ndim - 1 -i);
//   }

//   size_t size = total_elements;
//   if (size == 0)
//     return;
//   dim3 blocks, threads;
//   threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
//   blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
//   HT_DISPATCH_FLOATING_TYPES(
//     in_arr->dtype(), spec_t, "CauculateGradCuda", [&]() {
//       spec_t* ds = NULL;
//       DataPtr ds_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideC * sizeof(spec_t));
//       ds = (spec_t*) ds_ptr.ptr;

//       spec_t* db = NULL;
//       DataPtr db_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideC * sizeof(spec_t));
//       db = (spec_t*) db_ptr.ptr;

//       spec_t* dy_mul_x = NULL;
//       DataPtr dy_mul_x_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideA * sizeof(spec_t));
//       dy_mul_x = (spec_t*) dy_mul_x_ptr.ptr;

//       DataPtr gscale_ptr = AllocFromMemoryPool(out_grads->device(), temp_strideA * sizeof(spec_t));
//       spec_t* gscale = (spec_t*) gscale_ptr.ptr;

//       DataPtr workspace_ptr;
//       workspace_ptr = in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16 
//                                          ? AllocFromMemoryPool(out_grads->device(), temp_strideA * sizeof(float))
//                                          : AllocFromMemoryPool(out_grads->device(), temp_strideA * sizeof(spec_t));
//       spec_t* workspace = (spec_t*) workspace_ptr.ptr;

//       spec_t one = 1.0f;
//       spec_t zero = 0.0f;

//       float one_f = 1.0f;
//       float zero_f = 0.0f;

//       hipdnnReduceTensorDescriptor_t rtd;
//       CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
//       CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
//         rtd, HIPDNN_REDUCE_TENSOR_ADD, in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16 ? HIPDNN_DATA_FLOAT : datatype, HIPDNN_PROPAGATE_NAN,
//         HIPDNN_REDUCE_TENSOR_NO_INDICES, indicetype));

//       hipdnnTensorDescriptor_t adesc;
//       hipdnnTensorDescriptor_t bdesc;
//       hipdnnTensorDescriptor_t cdesc;
//       CUDNN_CALL(hipdnnCreateTensorDescriptor(&adesc));
//       CUDNN_CALL(hipdnnCreateTensorDescriptor(&bdesc));
//       CUDNN_CALL(hipdnnCreateTensorDescriptor(&cdesc));

//       CUDNN_CALL(
//         hipdnnSetTensorNdDescriptor(adesc, datatype, ndim, dimA, strideA));
//       CUDNN_CALL(
//         hipdnnSetTensorNdDescriptor(bdesc, datatype, ndim, dimB, strideB));
//       CUDNN_CALL(
//         hipdnnSetTensorNdDescriptor(cdesc, datatype, ndim, dimC, strideC));

      
//       if (in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16) {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(float), &one_f,
//           adesc, (const void*) out_grads->data_ptr<void>(), &zero_f, bdesc,
//           (void*) grad_bias->data_ptr<void>()));
//       }
//       else {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
//           adesc, (const void*) out_grads->data_ptr<void>(), &zero, bdesc,
//           (void*) grad_bias->data_ptr<void>()));
//       }

//       calculate_gscale<spec_t><<<blocks, threads, 0, hip_stream>>>(
//         out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
//         mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
//         gscale, eps, lastdim, (size_t) temp_strideA);
      

//       if (in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16) {  
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(float), &one_f,
//           adesc, (const void*) gscale, &zero_f, bdesc,
//           (void*) grad_scale->data_ptr<void>())); 
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(float), &one_f,
//           adesc, (const void*) out_grads->data_ptr<void>(), &zero_f, cdesc,
//           (void*) db));  
//       }   
//       else {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
//           adesc, (const void*) gscale, &zero, bdesc,
//           (void*) grad_scale->data_ptr<void>())); 
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
//           adesc, (const void*) out_grads->data_ptr<void>(), &zero, cdesc,
//           (void*) db));  
//       } 

//       auto op = kmultiplies<spec_t, spec_t>();

//       binary_elewise_kernel<spec_t, spec_t><<<blocks, threads, 0, hip_stream>>>(
//         out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
//         size, op, dy_mul_x);
      
//       if (in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16) {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(float), &one_f,
//           adesc, (const void*) dy_mul_x, &zero_f, cdesc,
//           (void*) ds));  
//       }
//       else {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
//           adesc, (const void*) dy_mul_x, &zero, cdesc,
//           (void*) ds));  
//       }
        

//       calculate_grad_kernel_layer<spec_t><<<blocks, threads, 0, hip_stream>>>(
//         out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
//         mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
//         ds, db,
//         grad_arr->data_ptr<spec_t>(), lastdim, eps, size);
//       FreeToMemoryPool(ds_ptr);
//       FreeToMemoryPool(db_ptr);
//       FreeToMemoryPool(dy_mul_x_ptr);
//       FreeToMemoryPool(gscale_ptr);
//       FreeToMemoryPool(workspace_ptr);
//     });
//     free(dimA);
//     free(strideA);
//     free(dimB);
//     free(strideB);
//     free(dimC);
//     free(strideC);
// }

void LayerNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
                           const NDArray& ln_scale, NDArray& grad_arr,
                           NDArray& grad_scale, NDArray& grad_bias,
                           const NDArray& mean_arr, const NDArray& var_arr,
                           int64_t reduce_dims, float eps, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(out_grads);
  HT_ASSERT_SAME_DEVICE(out_grads, ln_scale);
  HT_ASSERT_SAME_DEVICE(out_grads, in_arr);
  HT_ASSERT_SAME_DEVICE(out_grads, mean_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, var_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, grad_scale);
  HT_ASSERT_SAME_DEVICE(out_grads, grad_arr);
  HT_ASSERT_SAME_DEVICE(out_grads, grad_bias);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  int ndim = out_grads->ndim();
//   HT_ASSERT(ndim == 4);
  size_t total_elements = 1;

  int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);

  HTAxes reduce_axes_before = {}, reduce_axes_after = {};
  for (int i = 0; i < ndim; ++i) {
    if (i < ndim - reduce_dims)
      reduce_axes_before.emplace_back(i);
    else
      reduce_axes_after.emplace_back(i);
  }

  for (int i = 0; i < ndim; ++i)
    total_elements *= out_grads->shape(i);
  int lastdim = 1;
  for (size_t i = 0; i < reduce_dims; ++i) {
    lastdim *= out_grads->shape(ndim - 1 -i);
  }

  size_t size = total_elements;
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "CauculateGradCuda", [&]() {
    

      NDArray grad_bias_ = NDArray::sum(out_grads, reduce_axes_before, true, stream.stream_index());

      NDArray gscale_ = NDArray::empty_like(in_arr);

      calculate_gscale<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
        gscale_->data_ptr<spec_t>(), eps, lastdim, in_arr->numel());
      
      NDArray::sum(gscale_, reduce_axes_before, true, stream.stream_index(), grad_scale);

      NDArray db_ = NDArray::sum(out_grads, reduce_axes_after, true, stream.stream_index());

      NDArray dy_mul_x_ = NDArray::mul(out_grads, in_arr, stream.stream_index());

      NDArray ds_ = NDArray::sum(dy_mul_x_, reduce_axes_after, true, stream.stream_index());

      calculate_grad_kernel_layer<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
        ds_->data_ptr<spec_t>(), db_->data_ptr<spec_t>(),
        grad_arr->data_ptr<spec_t>(), lastdim, eps, size);
    });
}

} // namespace impl
} // namespace hetu
