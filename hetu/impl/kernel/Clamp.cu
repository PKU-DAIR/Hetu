#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void clamp_kernel(const spec_t* input, spec_t min_val, spec_t max_val, 
                             size_t size, spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  if (input[idx] < min_val)
    output[idx] = min_val;
  else if (input[idx] > max_val) {
    output[idx] = max_val;
  }
  else 
    output[idx] = input[idx];
}

template <typename spec_t>
__global__ void clamp_elewise_kernel(const spec_t* input, const spec_t* min_val, const spec_t* max_val, 
                                     size_t size, spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  if (input[idx] < min_val[idx])
    output[idx] = min_val[idx];
  else if (input[idx] > max_val[idx]) {
    output[idx] = max_val[idx];
  }
  else 
    output[idx] = input[idx];
}

void ClampCuda(const NDArray& input, double min_val, double max_val, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ClampCuda", [&]() {
      clamp_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), min_val, max_val, size, output->data_ptr<spec_t>());
    });
}

void ClampElewiseCuda(const NDArray& input, const NDArray& min_val, const NDArray& max_val, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ClampCuda", [&]() {
      clamp_elewise_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), min_val->data_ptr<spec_t>(), max_val->data_ptr<spec_t>(), 
        size, output->data_ptr<spec_t>());
    });
}

} // namespace impl
} // namespace hetu
