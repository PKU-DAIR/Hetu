#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void rangemask_kernel(const spec_t* input, int64_t min, 
                                 int64_t max, int64_t* output, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = (static_cast<int64_t>(input[idx]) >= min) && (static_cast<int64_t>(input[idx]) <= max) ? 0 : 1;
}

void RangeMaskCuda(const NDArray& input, int64_t min, int64_t max,
                  NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = input->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "RangeMaskCuda", [&]() {
      rangemask_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), min, max, output->data_ptr<int64_t>(), size);
    });
}

} // namespace impl
} // namespace hetu
