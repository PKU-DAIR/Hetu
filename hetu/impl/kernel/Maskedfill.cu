#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void maskedfill_kernel(const spec_t* input, const int64_t* mask, 
                                spec_t val, spec_t* output, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  bool mask_bit = bool(mask[idx]);
  output[idx] = mask_bit ? val : input[idx];
}

void MaskedfillCuda(const NDArray& input, const NDArray& mask,
                  double val, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, mask);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = input->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "MaskfillCuda", [&]() {
      maskedfill_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), mask->data_ptr<int64_t>(),
        static_cast<spec_t>(val), output->data_ptr<spec_t>(), size);
    });
}

} // namespace impl
} // namespace hetu
