#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include <chrono>

namespace hetu {
namespace impl {

void Conv3dCuda(const NDArray& input_x, const NDArray& input_f, NDArray& output,
                const int padding_d, const int padding_h, const int padding_w,
                const int stride_d, const int stride_h, const int stride_w, 
                const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_x);
  HT_ASSERT_SAME_DEVICE(input_x, input_f);
  HT_ASSERT_SAME_DEVICE(input_x, output);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_x->dtype());

  int inputDims[5] = {input_x->shape(0), input_x->shape(1), input_x->shape(2), input_x->shape(3), input_x->shape(4)};
  int inputStrides[5] = {input_x->stride(0), input_x->stride(1), input_x->stride(2), input_x->stride(3), input_x->stride(4)};

  int filterDims[5] = {input_f->shape(0), input_f->shape(1), input_f->shape(2), input_f->shape(3), input_f->shape(4)};
  int filterStrides[5] = {input_f->stride(0), input_f->stride(1), input_f->stride(2), input_f->stride(3), input_f->stride(4)}; 

  int outputDims[5] = {output->shape(0), output->shape(1), output->shape(2), output->shape(3), output->shape(4)};
  int outputStrides[5] = {output->stride(0), output->stride(1), output->stride(2), output->stride(3), output->stride(4)};


  // input
  hipdnnTensorDescriptor_t input_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(input_desc, datatype, 5, inputDims, inputStrides));

  // filter
  hipdnnFilterDescriptor_t filter_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_desc, datatype,
                                        HIPDNN_TENSOR_NCHW, 5, filterDims));

  // output
  hipdnnTensorDescriptor_t out_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(out_desc, datatype, 5, outputDims, outputStrides));

  // convolution
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  int conv_padding[3] = {padding_d, padding_h, padding_w};
  int conv_stride[3] = {stride_d, stride_h, stride_w};
  int conv_dilation[3] = {1, 1, 1};
  CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_desc, 3, conv_padding, conv_stride, conv_dilation, HIPDNN_CROSS_CORRELATION, datatype));

  if (input_x->dtype() == DataType::FLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));


  // algorithm
  hipdnnConvolutionFwdAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // workaround here
  // TODO: using hipdnnFindConvolutionForwardAlgorithm in CuDNN 8 instead
  int return_algo_cnt = HIPDNN_CONVOLUTION_FWD_ALGO_COUNT;
  hipdnnConvolutionFwdAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
  CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(
    handle, input_desc, filter_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_ALGO_COUNT, &return_algo_cnt, perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
      handle, input_desc, filter_desc, conv_desc, out_desc,
      perf_results[i].algo, &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
      handle, input_desc, filter_desc, conv_desc, out_desc, 
      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
#endif

  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    handle, input_desc, filter_desc, conv_desc, out_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_x->device(), kInt8, stream.stream_index());
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_x->dtype(), spec_t, "Conv3dCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0f;
      spec_t beta = 0.0f;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionForward(handle, &alpha_f, input_desc, input_x->data_ptr<spec_t>(),
                                           filter_desc, input_f->data_ptr<spec_t>(), conv_desc,
                                           algo, workspace_ptr, workspace_size, &beta_f,
                                           out_desc, output->data_ptr<spec_t>()));
      } else {
        CUDNN_CALL(hipdnnConvolutionForward(handle, &alpha, input_desc, input_x->data_ptr<spec_t>(),
                                           filter_desc, input_f->data_ptr<spec_t>(), conv_desc,
                                           algo, workspace_ptr, workspace_size, &beta,
                                           out_desc, output->data_ptr<spec_t>()));
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    });

  NDArray::MarkUsedBy({input_x, input_f, output, workspace}, stream);
  return;
}

void Conv3dGradientofFilterCuda(const NDArray& input_x,
                                const NDArray& gradient_y, NDArray& gradient_f,
                                const int padding_d, const int padding_h, const int padding_w,
                                const int stride_d, const int stride_h, const int stride_w,
                                const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_x);
  HT_ASSERT_SAME_DEVICE(input_x, gradient_y);
  HT_ASSERT_SAME_DEVICE(input_x, gradient_f);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_x->dtype());

  // input


  int input_dims[5] = {input_x->shape(0), input_x->shape(1), input_x->shape(2), input_x->shape(3), input_x->shape(4)};
  int input_strides[5] = {input_x->stride(0), input_x->stride(1), input_x->stride(2), input_x->stride(3), input_x->stride(4)};

  int dy_dims[5] = {gradient_y->shape(0), gradient_y->shape(1), gradient_y->shape(2), gradient_y->shape(3), gradient_y->shape(4)};
  int dy_strides[5] = {gradient_y->stride(0), gradient_y->stride(1), gradient_y->stride(2), gradient_y->stride(3), gradient_y->stride(4)};

  int df_dims[5] = {gradient_f->shape(0), gradient_f->shape(1), gradient_f->shape(2), gradient_f->shape(3), gradient_f->shape(4)};
  int df_strides[5] = {gradient_f->stride(0), gradient_f->stride(1), gradient_f->stride(2), gradient_f->stride(3), gradient_f->stride(4)};

  // input
  hipdnnTensorDescriptor_t input_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(input_desc, datatype, 5, input_dims, input_strides));

  // dy
  hipdnnTensorDescriptor_t dy_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(dy_desc, datatype, 5, dy_dims, dy_strides));


  // dw
  hipdnnFilterDescriptor_t df_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&df_desc));
  CUDNN_CALL(hipdnnSetFilterNdDescriptor(df_desc, datatype, HIPDNN_TENSOR_NCHW, 5, df_dims));


  // conv3d
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  int conv_padding[3] = {padding_d, padding_h, padding_w};
  int conv_stride[3] = {stride_d, stride_h, stride_w};
  int conv_dilation[3] = {1, 1, 1};
  CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_desc, 3, conv_padding, conv_stride, conv_dilation, HIPDNN_CROSS_CORRELATION, datatype));

  if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));

  // algo
  hipdnnConvolutionBwdFilterAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // TODO: using hipdnnFindConvolutionBackwardFilterAlgorithm in CuDNN 8
  // instead algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT;
  int return_algo_cnt = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT;
  hipdnnConvolutionBwdFilterAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT];
  CUDNN_CALL(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
    handle, input_desc, dy_desc, conv_desc, df_desc,
    HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT, &return_algo_cnt,
    perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
      handle, input_desc, dy_desc, conv_desc, df_desc, perf_results[i].algo,
      &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(
      handle, input_desc, dy_desc, conv_desc, df_desc,
      HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algo));
#endif
  CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
    handle, input_desc, dy_desc, conv_desc, df_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_x->device(), kInt8, stream.stream_index());
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_x->dtype(), spec_t, "Conv2dGradientofFilterCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionBackwardFilter(
          handle, &alpha_f, input_desc, input_x->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta_f, df_desc, gradient_f->data_ptr<spec_t>()));
      } else {
        CUDNN_CALL(hipdnnConvolutionBackwardFilter(
          handle, &alpha, input_desc, input_x->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta, df_desc, gradient_f->data_ptr<spec_t>()));
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(df_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    });
  NDArray::MarkUsedBy({input_x, gradient_y, gradient_f, workspace}, stream);
}


void Conv3dGradientofDataCuda(const NDArray& input_f, const NDArray& gradient_y,
                              NDArray& gradient_x, const int padding_d, const int padding_h,
                              const int padding_w, const int stride_d, const int stride_h,
                              const int stride_w, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_f);
  HT_ASSERT_SAME_DEVICE(input_f, gradient_y);
  HT_ASSERT_SAME_DEVICE(input_f, gradient_x);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_f->dtype());

  // filter
  int filter_dim[5] = {input_f->shape(0), input_f->shape(1), input_f->shape(2), input_f->shape(3), input_f->shape(4)};
  int filter_strides[5] = {input_f->stride(0), input_f->stride(1), input_f->stride(2), input_f->stride(3), input_f->stride(4)};

  // dy
  int dy_dims[5] = {gradient_y->shape(0), gradient_y->shape(1), gradient_y->shape(2), gradient_y->shape(3), gradient_y->shape(4)};
  int dy_strides[5] = {gradient_y->stride(0), gradient_y->stride(1), gradient_y->stride(2), gradient_y->stride(3), gradient_y->stride(4)};
  
  // dx
  int dx_dims[5] = {gradient_x->shape(0), gradient_x->shape(1), gradient_x->shape(2), gradient_x->shape(3), gradient_x->shape(4)};
  int dx_strides[5] = {gradient_x->stride(0), gradient_x->stride(1), gradient_x->stride(2), gradient_x->stride(3), gradient_x->stride(4)};

  // filter
  hipdnnFilterDescriptor_t filter_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnSetFilterNdDescriptor(filter_desc, datatype, HIPDNN_TENSOR_NCHW, 5, filter_dim));

  // dy
  hipdnnTensorDescriptor_t dy_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(dy_desc, datatype, 5, dy_dims, dy_strides));


  // dx
  hipdnnTensorDescriptor_t dx_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&dx_desc));
  CUDNN_CALL(hipdnnSetTensorNdDescriptor(dx_desc, datatype, 5, dx_dims, dx_strides));


  // conv3d
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  int conv_padding[3] = {padding_d, padding_h, padding_w};
  int conv_stride[3] = {stride_d, stride_h, stride_w};
  int conv_dilation[3] = {1, 1, 1};
  CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(conv_desc, 3, conv_padding, conv_stride, conv_dilation, HIPDNN_CROSS_CORRELATION, datatype));

  if (input_f->dtype() == DataType::FLOAT16 || input_f->dtype() == DataType::BFLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));

  // algo
  hipdnnConvolutionBwdDataAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // TODO: using hipdnnFindConvolutionBackwardDataAlgorithm in CuDNN 8
  // instead
  int return_algo_cnt = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM;
  hipdnnConvolutionBwdDataAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM];
  CUDNN_CALL(cudnnGetConvolutionBackwardDataAlgorithm_v7(
    handle, filter_desc, dy_desc, conv_desc, dx_desc,
    HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM, &return_algo_cnt, perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
      handle, filter_desc, dy_desc, conv_desc, dx_desc,
      perf_results[i].algo, &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(
      handle, filter_desc, dy_desc, conv_desc, dx_desc,
      HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algo));
#endif
  CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
    handle, filter_desc, dy_desc, conv_desc, dx_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_f->device(), kInt8, stream.stream_index());
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_f->dtype(), spec_t, "Conv2dGradientofDataCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input_f->dtype() == DataType::FLOAT16 || input_f->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionBackwardData(
          handle, &alpha_f, filter_desc, input_f->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta_f, dx_desc, gradient_x->data_ptr<spec_t>()));
      } else {
        CUDNN_CALL(hipdnnConvolutionBackwardData(
          handle, &alpha, filter_desc, input_f->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta, dx_desc, gradient_x->data_ptr<spec_t>()));        
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(dx_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    });
  NDArray::MarkUsedBy({input_f, gradient_y, gradient_x, workspace}, stream);
}

} // namespace impl
} // namespace hetu
