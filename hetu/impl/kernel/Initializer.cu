#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/cuda/CUDARand.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/random/CUDARandomState.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void init_normal_kernel(spec_t* arr, size_t size, spec_t mean,
                                   spec_t stddev, CUDARandomState rand_state) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(rand_state.seed, idx, rand_state.offset, &state);
  arr[idx] = hiprand_normal(&state) * stddev + mean;
}

template <typename spec_t>
__global__ void init_uniform_kernel(spec_t* arr, size_t size, spec_t lb,
                                    spec_t ub, CUDARandomState rand_state) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(rand_state.seed, idx, rand_state.offset, &state);
  arr[idx] = hiprand_uniform(&state) * (ub - lb) + lb;
}

template <typename spec_t>
__global__ void init_truncated_normal_kernel(spec_t* arr, size_t size,
                                             spec_t mean, spec_t stddev,
                                             spec_t lb, spec_t ub,
                                             CUDARandomState rand_state) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(rand_state.seed, idx, rand_state.offset, &state);
  do {
    arr[idx] = hiprand_normal(&state) * stddev + mean;
  } while (arr[idx] < lb || arr[idx] > ub);
}

void NormalInitsCuda(NDArray& data, double mean, double stddev, uint64_t seed,
                     const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(data);
  size_t size = data->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    data->dtype(), spec_t, "NormalInitsCuda", [&]() {
      init_normal_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        data->data_ptr<spec_t>(), size, static_cast<spec_t>(mean),
        static_cast<spec_t>(stddev),
        GetCUDARandomState(hip_stream.device_id(), seed, 4));
    });
  NDArray::MarkUsedBy({data}, stream);
}

void UniformInitsCuda(NDArray& data, double lb, double ub, uint64_t seed,
                      const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(data);
  HT_ASSERT(lb < ub) << "Invalid range for uniform random init: "
                     << "[" << lb << ", " << ub << ").";
  size_t size = data->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    data->dtype(), spec_t, "UniformInitCuda", [&]() {
      init_uniform_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        data->data_ptr<spec_t>(), size, static_cast<spec_t>(lb),
        static_cast<spec_t>(ub),
        GetCUDARandomState(hip_stream.device_id(), seed, 4));
    });
  NDArray::MarkUsedBy({data}, stream);
}

void TruncatedNormalInitsCuda(NDArray& data, double mean, double stddev,
                              double lb, double ub, uint64_t seed,
                              const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(data);
  size_t size = data->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    data->dtype(), spec_t, "TruncatedNormalInitsCuda", [&]() {
      init_truncated_normal_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        data->data_ptr<spec_t>(), size, static_cast<spec_t>(mean),
        static_cast<spec_t>(stddev), static_cast<spec_t>(lb),
        static_cast<spec_t>(ub),
        GetCUDARandomState(hip_stream.device_id(), seed, 32));
    });
  NDArray::MarkUsedBy({data}, stream);
}

} // namespace impl
} // namespace hetu
