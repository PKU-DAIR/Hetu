#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void array_set_kernel(spec_t* arr, spec_t value, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    arr[idx] = value;
}

void ArraySetCuda(NDArray& data, double value, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(data);
  size_t size = data->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    data->dtype(), spec_t, "ArraySetCuda", [&]() {
      array_set_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        data->data_ptr<spec_t>(), static_cast<spec_t>(value), size);
    });
}

} // namespace impl
} // namespace hetu
