#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void bool_kernel(const spec_t* input, size_t size, bool* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  if (input[idx] > 0)
    output[idx] = 1;
  else
    output[idx] = 0;
}

void BoolCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT(output->dtype() == DataType::BOOL);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "BoolCuda", [&]() {
      bool_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), size, output->data_ptr<bool>());
    });
}

} // namespace impl
} // namespace hetu
