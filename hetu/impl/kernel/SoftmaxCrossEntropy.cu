#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void softmax_cross_entropy_kernel(const spec_t* logsoftmax,
                                             const spec_t* label,
                                             spec_t* output, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = -logsoftmax[idx] * label[idx];
}

void SoftmaxCrossEntropyCuda(const NDArray& input, const NDArray& label,
                             NDArray& output, const Stream& stream) {
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());
  size_t indim = input->ndim();
  HT_ASSERT(indim == label->ndim() && indim == output->ndim() + 1)
    << "Indim is " << indim << ", Label dim is " << label->ndim()
    << ", Output dim is " << output->ndim();
  int n_ = 1;
  for (int i = 0; i < indim - 1; ++i) {
    n_ *= input->shape(i);
  }
  int c_ = input->shape(indim - 1);
  size_t size = n_ * c_;

  if (size == 0)
    return;

  hipdnnDataType_t datatype = to_cudnn_DataType(input->dtype());
  hipdnnIndicesType_t indicetype = to_cudnn_IndicidesType(input->dtype());

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "SoftmaxCrossEntropyCuda", [&]() {
      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      hipdnnTensorDescriptor_t desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, datatype,
                                            n_, c_, 1, 1));
      NDArray temp_ = NDArray::empty_like(input);

      if (input->dtype() == DataType::FLOAT16 || input->dtype() == DataType::BFLOAT16) {
      CUDNN_CALL(hipdnnSoftmaxForward(
          handle, HIPDNN_SOFTMAX_LOG, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha_f, desc,
          (const void*) input->data_ptr<spec_t>(), &beta_f, desc, (void*)temp_->data_ptr<spec_t>()));     
      }
      else {
      CUDNN_CALL(hipdnnSoftmaxForward(
          handle, HIPDNN_SOFTMAX_LOG, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, desc,
          (const void*) input->data_ptr<spec_t>(), &beta, desc, (void*)temp_->data_ptr<spec_t>()));             
      }   


      softmax_cross_entropy_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        temp_->data_ptr<spec_t>(), label->data_ptr<spec_t>(),
        temp_->data_ptr<spec_t>(), size);
      NDArray::sum(temp_, {1}, false, stream.stream_index(), output);
    });
  NDArray::MarkUsedBy({input, label, output}, stream);
}

template <typename spec_t>
__global__ void softmax_cross_entropy_gradient_kernel(
  const spec_t* pred, const spec_t* y_, const spec_t* grad_data,
  spec_t* output_data, int last_dim, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output_data[idx] = (pred[idx] - y_[idx]) * grad_data[idx / last_dim];
}

void SoftmaxCrossEntropyGradientCuda(const NDArray& input_y,
                                     const NDArray& label, const NDArray& grad,
                                     NDArray& output, const Stream& stream) {
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());
  size_t indim = input_y->ndim();
  HT_ASSERT(indim == label->ndim() && indim == output->ndim() &&
            indim == grad->ndim() + 1)
    << "Indim is " << indim << ", Label dim is " << label->ndim()
    << ", Output dim is " << output->ndim();
  int n_ = 1;
  for (int i = 0; i < indim - 1; ++i) {
    n_ *= input_y->shape(i);
  }
  int c_ = input_y->shape(indim - 1);
  size_t size = n_ * c_;

  hipdnnDataType_t datatype = to_cudnn_DataType(input_y->dtype());

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);

  int64_t workspace_size;
  if (input_y->dtype() == DataType::FLOAT16 || input_y->dtype() == DataType::BFLOAT16) {
    workspace_size = size * sizeof(float);
  } else {
    workspace_size = size * DataType2Size(input_y->dtype());
  }
  auto workspace_arr =
    NDArray::empty({workspace_size}, grad->device(), kInt8, stream.stream_index());

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_y->dtype(), spec_t, "SoftmaxCrossEntropyCuda", [&]() {
      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      hipdnnTensorDescriptor_t desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, datatype,
                                            n_, c_, 1, 1));
      if (input_y->dtype() == DataType::FLOAT16 || input_y->dtype() == DataType::BFLOAT16) {
      CUDNN_CALL(hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha_f,
        desc, input_y->data_ptr<spec_t>(), &beta_f, desc, 
        workspace_arr->raw_data_ptr()));
      } else {
      CUDNN_CALL(hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha,
        desc, input_y->data_ptr<spec_t>(), &beta, desc, 
        workspace_arr->raw_data_ptr()));        
      }

      softmax_cross_entropy_gradient_kernel<spec_t>
        <<<blocks, threads, 0, hip_stream>>>(
          (const spec_t*) workspace_arr->raw_data_ptr(), label->data_ptr<spec_t>(),
          grad->data_ptr<spec_t>(), output->data_ptr<spec_t>(), c_, size);

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
    });
  NDArray::MarkUsedBy({input_y, label, grad, workspace_arr}, stream);
}

} // namespace impl
} // namespace hetu
