#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void softmax_cross_entropy_kernel(const spec_t* logsoftmax,
                                             const spec_t* label,
                                             spec_t* output, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = -logsoftmax[idx] * label[idx];
}

void SoftmaxCrossEntropyCuda(const NDArray& input, const NDArray& label,
                             NDArray& output, const Stream& stream) {
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());
  size_t indim = input->ndim();
  HT_ASSERT(indim == label->ndim() && indim == output->ndim() + 1)
    << "Indim is " << indim << ", Label dim is " << label->ndim()
    << ", Output dim is " << output->ndim();
  int n_ = 1;
  for (int i = 0; i < indim - 1; ++i) {
    n_ *= input->shape(i);
  }
  int c_ = input->shape(indim - 1);
  size_t size = n_ * c_;

  if (size == 0)
    return;

  int dev_id = hip_stream.device_id();
  hipdnnDataType_t datatype;
  hipdnnIndicesType_t indicetype;
  if (input->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
    indicetype = HIPDNN_32BIT_INDICES;
  } else if (input->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
    indicetype = HIPDNN_64BIT_INDICES;
  }

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "SoftmaxCrossEntropyCuda", [&]() {
      spec_t alpha = 1.0;
      spec_t beta = 0.0;
      hipdnnTensorDescriptor_t desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, datatype,
                                            n_, c_, 1, 1));
      DataPtr temp_data_ptr =
        AllocFromMemoryPool(input->device(), size * sizeof(spec_t));
      void* temp_data = temp_data_ptr.ptr;

      CUDNN_CALL(hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_LOG, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, desc,
        (const void*) input->data_ptr<spec_t>(), &beta, desc, temp_data));

      softmax_cross_entropy_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        (const spec_t*) temp_data, label->data_ptr<spec_t>(),
        (spec_t*) temp_data, size);

      hipdnnReduceTensorDescriptor_t rtd;
      CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
      CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
        rtd, HIPDNN_REDUCE_TENSOR_ADD, datatype, HIPDNN_PROPAGATE_NAN,
        HIPDNN_REDUCE_TENSOR_NO_INDICES, indicetype));

      hipdnnTensorDescriptor_t new_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&new_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(new_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, n_, 1, 1, 1));
      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, temp_data, size * sizeof(spec_t), &alpha, desc,
        (const void*) temp_data, &beta, new_desc, output->data_ptr<spec_t>()));

      CUDNN_CALL(hipdnnDestroyReduceTensorDescriptor(rtd));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(new_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
      FreeToMemoryPool(temp_data_ptr);
    });
}

template <typename spec_t>
__global__ void softmax_cross_entropy_gradient_kernel(
  const spec_t* pred, const spec_t* y_, const spec_t* grad_data,
  spec_t* output_data, int last_dim, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output_data[idx] = (pred[idx] - y_[idx]) * grad_data[idx / last_dim];
}

void SoftmaxCrossEntropyGradientCuda(const NDArray& input_y,
                                     const NDArray& label, const NDArray& grad,
                                     NDArray& output, const Stream& stream) {
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());
  size_t indim = input_y->ndim();
  HT_ASSERT(indim == label->ndim() && indim == output->ndim() &&
            indim == grad->ndim() + 1)
    << "Indim is " << indim << ", Label dim is " << label->ndim()
    << ", Output dim is " << output->ndim();
  int n_ = 1;
  for (int i = 0; i < indim - 1; ++i) {
    n_ *= input_y->shape(i);
  }
  int c_ = input_y->shape(indim - 1);
  size_t size = n_ * c_;

  hipdnnDataType_t datatype;
  if (input_y->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (input_y->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  }

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_y->dtype(), spec_t, "SoftmaxCrossEntropyCuda", [&]() {
      int dev_id = hip_stream.device_id();

      DataPtr temp_data_ptr =
        AllocFromMemoryPool(grad->device(), size * sizeof(spec_t));
      void* temp_data = temp_data_ptr.ptr;

      spec_t alpha = 1.0;
      spec_t beta = 0.0;
      hipdnnTensorDescriptor_t desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, datatype,
                                            n_, c_, 1, 1));
      CUDNN_CALL(hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha,
        desc, input_y->data_ptr<spec_t>(), &beta, desc, temp_data));

      softmax_cross_entropy_gradient_kernel<spec_t>
        <<<blocks, threads, 0, hip_stream>>>(
          (const spec_t*) temp_data, label->data_ptr<spec_t>(),
          grad->data_ptr<spec_t>(), output->data_ptr<spec_t>(), c_, size);

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
      FreeToMemoryPool(temp_data_ptr);
    });
}

} // namespace impl
} // namespace hetu
