#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

void BatchNormCuda(const NDArray& input_X, const NDArray& bn_scale,
                   const NDArray& bn_bias, NDArray& output_Y, double momentum,
                   double eps, NDArray& running_mean, NDArray& running_var,
                   NDArray& save_mean, NDArray& save_var,
                   const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_X);
  HT_ASSERT_SAME_DEVICE(input_X, bn_scale);
  HT_ASSERT_SAME_DEVICE(input_X, bn_bias);
  HT_ASSERT_SAME_DEVICE(input_X, output_Y);
  HT_ASSERT_SAME_DEVICE(input_X, running_mean);
  HT_ASSERT_SAME_DEVICE(input_X, running_var);
  HT_ASSERT_SAME_DEVICE(input_X, save_mean);
  HT_ASSERT_SAME_DEVICE(input_X, save_var);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype;
  if (input_X->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (input_X->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  }

  // input
  size_t input_N = input_X->shape(0);
  size_t input_C = input_X->shape(1);
  size_t input_H = input_X->shape(2);
  size_t input_W = input_X->shape(3);

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_X->dtype(), spec_t, "BatchNormCuda", [&]() {
      const spec_t* input_data = (const spec_t*) (input_X->data_ptr<spec_t>());
      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // output
      spec_t* output_data = (spec_t*) (output_Y->data_ptr<spec_t>());
      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // bn parameter descriptor
      hipdnnTensorDescriptor_t bnScaleBiasMeanVar_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVar_desc));
      CUDNN_CALL(
        hipdnnDeriveBNTensorDescriptor(bnScaleBiasMeanVar_desc, input_desc,
                                      HIPDNN_BATCHNORM_SPATIAL)); // after conv
      // bn parameter
      const spec_t* bn_scale_data =
        (const spec_t*) (bn_scale->data_ptr<spec_t>());
      const spec_t* bn_bias_data =
        (const spec_t*) (bn_bias->data_ptr<spec_t>());
      void* running_mean_arr = running_mean->data_ptr<void>();
      void* running_var_arr = running_var->data_ptr<void>();
      void* save_mean_arr = save_mean->data_ptr<void>();
      void* save_var_arr = save_var->data_ptr<void>();
      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      CUDNN_CALL(hipdnnBatchNormalizationForwardTraining(
        handle, HIPDNN_BATCHNORM_SPATIAL, &alpha, &beta, input_desc, input_data,
        output_desc, output_data, bnScaleBiasMeanVar_desc, bn_scale_data,
        bn_bias_data, momentum, running_mean_arr, running_var_arr, eps,
        save_mean_arr, save_var_arr));

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(bnScaleBiasMeanVar_desc));
    });
  return;
}

void BatchNormGradientCuda(const NDArray& gradient_Y, const NDArray& input_X,
                           const NDArray& bn_scale, NDArray& gradient_X,
                           NDArray& gradient_bn_scale,
                           NDArray& gradient_bn_bias, double eps,
                           NDArray& save_mean, NDArray& save_var,
                           const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(gradient_Y);
  HT_ASSERT_SAME_DEVICE(gradient_Y, input_X);
  HT_ASSERT_SAME_DEVICE(gradient_Y, bn_scale);
  HT_ASSERT_SAME_DEVICE(gradient_Y, gradient_X);
  HT_ASSERT_SAME_DEVICE(gradient_Y, gradient_bn_scale);
  HT_ASSERT_SAME_DEVICE(gradient_Y, gradient_bn_bias);
  HT_ASSERT_SAME_DEVICE(gradient_Y, save_mean);
  HT_ASSERT_SAME_DEVICE(gradient_Y, save_var);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype;
  if (input_X->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (input_X->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  }

  // input
  size_t input_N = input_X->shape(0);
  size_t input_C = input_X->shape(1);
  size_t input_H = input_X->shape(2);
  size_t input_W = input_X->shape(3);

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_X->dtype(), spec_t, "BatchNormGradientCuda", [&]() {
      const spec_t* input_data = (const spec_t*) (input_X->data_ptr<spec_t>());
      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // output
      const spec_t* gradient_y_data =
        (const spec_t*) (gradient_Y->data_ptr<spec_t>());
      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // bn parameter descriptor
      hipdnnTensorDescriptor_t bnScaleBiasMeanVar_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVar_desc));
      CUDNN_CALL(
        hipdnnDeriveBNTensorDescriptor(bnScaleBiasMeanVar_desc, input_desc,
                                      HIPDNN_BATCHNORM_SPATIAL)); // after conv

      const spec_t* bn_scale_data =
        (const spec_t*) (bn_scale->data_ptr<spec_t>());
      // x gradient
      spec_t* gradient_x_data = (spec_t*) (gradient_X->data_ptr<spec_t>());
      // bn gradient
      spec_t* gradient_bn_bias_data =
        (spec_t*) (gradient_bn_bias->data_ptr<spec_t>());
      spec_t* gradient_bn_scale_data =
        (spec_t*) (gradient_bn_scale->data_ptr<spec_t>());
      void* save_mean_arr = save_mean->data_ptr<void>();
      void* save_var_arr = save_var->data_ptr<void>();
      spec_t one = 1.0;
      spec_t zero = 0.0;

      CUDNN_CALL(hipdnnBatchNormalizationBackward(
        handle, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT, &one, &zero, &one, &zero,
        input_desc, input_data, output_desc, gradient_y_data, input_desc,
        gradient_x_data, bnScaleBiasMeanVar_desc, bn_scale_data,
        gradient_bn_scale_data, gradient_bn_bias_data, eps, save_mean_arr,
        save_var_arr));

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(bnScaleBiasMeanVar_desc));
    //   HT_LOG_INFO << gradient_X->shape() << " " << gradient_X->data_ptr<void>();
    });
}

} // namespace impl
} // namespace hetu
