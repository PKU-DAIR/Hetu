#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

void BatchNormCuda(const NDArray& input_X, const NDArray& bn_scale,
                   const NDArray& bn_bias, NDArray& output_Y, double momentum,
                   double eps, NDArray& running_mean, NDArray& running_var,
                   NDArray& save_mean, NDArray& save_var,
                   const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_X);
  HT_ASSERT_SAME_DEVICE(input_X, bn_scale);
  HT_ASSERT_SAME_DEVICE(input_X, bn_bias);
  HT_ASSERT_SAME_DEVICE(input_X, output_Y);
  HT_ASSERT_SAME_DEVICE(input_X, running_mean);
  HT_ASSERT_SAME_DEVICE(input_X, running_var);
  HT_ASSERT_SAME_DEVICE(input_X, save_mean);
  HT_ASSERT_SAME_DEVICE(input_X, save_var);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype;
  if (input_X->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (input_X->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  }

  // input
  size_t input_N = input_X->shape(0);
  size_t input_C = input_X->shape(1);
  size_t input_H = input_X->shape(2);
  size_t input_W = input_X->shape(3);

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_X->dtype(), spec_t, "BatchNormCuda", [&]() {
      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // bn parameter descriptor
      hipdnnTensorDescriptor_t bnScaleBiasMeanVar_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVar_desc));
      CUDNN_CALL(
        hipdnnDeriveBNTensorDescriptor(bnScaleBiasMeanVar_desc, input_desc,
                                      HIPDNN_BATCHNORM_SPATIAL)); // after conv

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      CUDNN_CALL(hipdnnBatchNormalizationForwardTraining(
        handle, HIPDNN_BATCHNORM_SPATIAL, &alpha, &beta, input_desc, input_X->data_ptr<spec_t>(),
        output_desc, output_Y->data_ptr<spec_t>(), bnScaleBiasMeanVar_desc, bn_scale->data_ptr<spec_t>(),
        bn_bias->data_ptr<spec_t>(), momentum, running_mean->data_ptr<void>(), running_var->data_ptr<void>(), eps,
        save_mean->data_ptr<void>(), save_var->data_ptr<void>()));

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(bnScaleBiasMeanVar_desc));
    });
  return;
}

void BatchNormGradientCuda(const NDArray& gradient_Y, const NDArray& input_X,
                           const NDArray& bn_scale, NDArray& gradient_X,
                           NDArray& gradient_bn_scale,
                           NDArray& gradient_bn_bias, double eps,
                           NDArray& save_mean, NDArray& save_var,
                           const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(gradient_Y);
  HT_ASSERT_SAME_DEVICE(gradient_Y, input_X);
  HT_ASSERT_SAME_DEVICE(gradient_Y, bn_scale);
  HT_ASSERT_SAME_DEVICE(gradient_Y, gradient_X);
  HT_ASSERT_SAME_DEVICE(gradient_Y, gradient_bn_scale);
  HT_ASSERT_SAME_DEVICE(gradient_Y, gradient_bn_bias);
  HT_ASSERT_SAME_DEVICE(gradient_Y, save_mean);
  HT_ASSERT_SAME_DEVICE(gradient_Y, save_var);

  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype;
  if (input_X->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
  } else if (input_X->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
  }

  // input
  size_t input_N = input_X->shape(0);
  size_t input_C = input_X->shape(1);
  size_t input_H = input_X->shape(2);
  size_t input_W = input_X->shape(3);

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_X->dtype(), spec_t, "BatchNormGradientCuda", [&]() {
      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));
      // bn parameter descriptor
      hipdnnTensorDescriptor_t bnScaleBiasMeanVar_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVar_desc));
      CUDNN_CALL(
        hipdnnDeriveBNTensorDescriptor(bnScaleBiasMeanVar_desc, input_desc,
                                      HIPDNN_BATCHNORM_SPATIAL)); // after conv

      spec_t one = 1.0;
      spec_t zero = 0.0;

      CUDNN_CALL(hipdnnBatchNormalizationBackward(
        handle, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT, &one, &zero, &one, &zero,
        input_desc, input_X->data_ptr<spec_t>(), output_desc, gradient_Y->data_ptr<spec_t>(), input_desc,
        gradient_X->data_ptr<spec_t>(), bnScaleBiasMeanVar_desc, bn_scale->data_ptr<spec_t>(),
        gradient_bn_scale->data_ptr<spec_t>(), gradient_bn_bias->data_ptr<spec_t>(), eps, 
        save_mean->data_ptr<void>(), save_var->data_ptr<void>()));

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(bnScaleBiasMeanVar_desc));
    });
}

} // namespace impl
} // namespace hetu
