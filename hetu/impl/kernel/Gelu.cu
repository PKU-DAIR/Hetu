#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"
#include "hetu/impl/utils/offset_calculator.cuh"
#include "hetu/impl/kernel/Vectorized.cuh"

#define SQRT_1_2  0.70710678118654757274f
#define pi 3.14159265358979323846f
#define e  2.71828182845904523536f

namespace hetu {
namespace impl {

void GeluCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = output->numel();
  if (size == 0)
    return;
  HT_DISPATCH_FLOATING_TYPES(
    input->dtype(), spec_t, "GeluCuda", [&]() {
      launch_loop_kernel<spec_t, spec_t>(input, output, size, stream,
                                         [=] __device__ (spec_t x) -> spec_t {
                                           return x * 0.5f *
                                              (1.0f + hetu::cuda::cuda_erf(x * SQRT_1_2));
                                         });
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

template <typename spec_t>
__global__ void gelu_gradient_kernel(const spec_t* input, const spec_t* output_grad,
                                     size_t size, spec_t* output,
                                     const OffsetCalculator* in_offset_calculator,
                                     const OffsetCalculator* out_grad_offset_calculator,
                                     const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto in_offset = in_offset_calculator->get(idx);
  auto out_grad_offset = out_grad_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  output[out_offset] = output_grad[out_grad_offset]*(0.5f + 0.5f * hetu::cuda::cuda_erf(input[in_offset] / hetu::cuda::cuda_sqrt(2.0)) + 
                     0.5f * input[in_offset]*(hetu::cuda::cuda_sqrt(2.0f) * 
                     hetu::cuda::cuda_exp(-0.5f * hetu::cuda::cuda_pow(input[in_offset], spec_t(2.0f))) / hetu::cuda::cuda_sqrt(pi)));
}

void GeluGradientCuda(const NDArray& input, const NDArray& output_grad,
                      NDArray& input_grad, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output_grad);
  HT_ASSERT_SAME_DEVICE(input, input_grad);
  HT_ASSERT_EXCHANGABLE(input, output_grad);
  HT_ASSERT_EXCHANGABLE(input, input_grad);

  size_t size = input_grad->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_grad_offset_calculator_arr,
          in_grad_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_grad_offset_calculator,
                   *in_grad_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_grad_offset_calculator_arr, out_grad_offset_calculator) = 
    AllocOffsetCalculator(output_grad, stream);
  std::tie(in_grad_offset_calculator_arr, in_grad_offset_calculator) = 
    AllocOffsetCalculator(input_grad, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "GeluGradientCuda", [&]() {
      gelu_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output_grad->data_ptr<spec_t>(), size,
        input_grad->data_ptr<spec_t>(), in_offset_calculator,
        out_grad_offset_calculator, in_grad_offset_calculator);
  });
  NDArray::MarkUsedBy({input, output_grad, input_grad, in_offset_calculator_arr,
                      out_grad_offset_calculator_arr, in_grad_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hetu
