#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/kernel/Binary.cuh"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_a_t, typename spec_b_t, typename Operator>
extern __global__ void binary_elewise_kernel(const spec_a_t* inputA, const spec_b_t* inputB,
                                             size_t size, Operator op, spec_a_t* output);

template <typename spec_t>
__global__ void minus_mean_n_square_kernel1(const spec_t* in_arr,
                                            const spec_t* mean, spec_t* out_arr,
                                            int last_2dim, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  spec_t temp = in_arr[idx] - mean[idx / last_2dim];
  out_arr[idx] = temp * temp;
}

template <typename spec_t>
__global__ void std_normal_transform(const spec_t* in_arr,
                                     const spec_t* mean_arr,
                                     const spec_t* var_arr, spec_t* out_arr,
                                     int last_2dim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / last_2dim;
  out_arr[idx] =
    (in_arr[idx] - mean_arr[mo_idx]) / hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

template <typename spec_t>
__global__ void instance_norm_kernel(const spec_t* x,  spec_t* y, spec_t* mean,
                                     spec_t* var, const float eps,
                                     const int last_dim) {
  __shared__ spec_t var_share;
  __shared__ spec_t mean_share;
  __shared__ spec_t shared_var[32];
  __shared__ spec_t shared_mean[32];

  int begin = blockIdx.x * last_dim + threadIdx.x;
  int end = (blockIdx.x + 1) * last_dim;

  spec_t mean_thread = 0, var_thread = 0;
  for (int i = begin; i < end; i += blockDim.x) {
    mean_thread += x[i];
    var_thread += (x[i] * x[i]);
  }

  hetu::cuda::BlockReduceSum(mean_thread, shared_mean);
  hetu::cuda::BlockReduceSum(var_thread, shared_var);
  if (threadIdx.x == 0) {
    mean[blockIdx.x] = mean_share = mean_thread / last_dim;
    var_share = var_thread / last_dim - mean_share * mean_share;
    if (double(var_share) < 0)
      var_share = 0;
    var[blockIdx.x] = var_share;
  }
  __syncthreads();

  mean_thread = mean_share;
  var_thread = var_share;
  spec_t tmp = 1.0f / sqrtf(var_thread + eps);
  for (int i = begin, j = threadIdx.x; i < end;
       i += blockDim.x, j += blockDim.x)
    y[i] = (x[i] - mean_thread) * tmp;
}

void InstanceNormCuda(const NDArray& in_arr, NDArray& mean_arr,
                      NDArray& var_arr, NDArray& out_arr, float eps,
                      const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(in_arr);
  HT_ASSERT_SAME_DEVICE(in_arr, mean_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, var_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, out_arr);   

  int ndim = in_arr->ndim();
  HT_ASSERT(ndim == 4);
  int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);
  int base_dim = in_arr->shape(0) * in_arr->shape(1);

  auto device_id = in_arr->device().index();
  hetu::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  dim3 blocks, threads;
  threads.x = (last_2dim >= 1024 ? 1024 : 64);
  blocks.x = base_dim;
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "InstanceNormCuda", [&]() {
      instance_norm_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        in_arr->data_ptr<spec_t>(), out_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(), 
        eps, last_2dim);
    });
  NDArray::MarkUsedBy({in_arr, mean_arr, var_arr, out_arr}, stream);
}

template <typename spec_t>
__global__ void calculate_grad_kernel(const spec_t* out_grads,
                                      const spec_t* in_arr,
                                      const spec_t* mean_arr,
                                      const spec_t* var_arr, 
                                      spec_t* ds, spec_t* dbias,
                                      spec_t* grad_arr,
                                      size_t last2dim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / last2dim;
  spec_t y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  spec_t tmp = (dbias[mo_idx] * mean_arr[mo_idx] - ds[mo_idx]) * (in_arr[idx] - mean_arr[mo_idx]) /
                (var_arr[mo_idx] + eps);
  grad_arr[idx] = out_grads[idx] / hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps) +
    ((tmp - dbias[mo_idx]) / (spec_t)last2dim) / 
    hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

template <>
__global__ void calculate_grad_kernel<float16>(const float16* out_grads,
                                      const float16* in_arr,
                                      const float16* mean_arr,
                                      const float16* var_arr, 
                                      float16* ds, float16* dbias,
                                      float16* grad_arr,
                                      size_t last2dim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / last2dim;
  float16 y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  float16 tmp = (dbias[mo_idx] * mean_arr[mo_idx] - ds[mo_idx]) * (in_arr[idx] - mean_arr[mo_idx]) /
                (var_arr[mo_idx] + eps);
  grad_arr[idx] = out_grads[idx] / hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps) +
    ((tmp - dbias[mo_idx]) / (float16)last2dim) / 
    hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

template <>
__global__ void calculate_grad_kernel<bfloat16>(const bfloat16* out_grads,
                                      const bfloat16* in_arr,
                                      const bfloat16* mean_arr,
                                      const bfloat16* var_arr, 
                                      bfloat16* ds, bfloat16* dbias,
                                      bfloat16* grad_arr,
                                      size_t last2dim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / last2dim;
  float16 y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  float16 tmp = (dbias[mo_idx] * mean_arr[mo_idx] - ds[mo_idx]) * (in_arr[idx] - mean_arr[mo_idx]) /
                (var_arr[mo_idx] + eps);
  grad_arr[idx] = out_grads[idx] / hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps) +
    ((tmp - dbias[mo_idx]) / (float16)last2dim) / 
    hetu::cuda::cuda_sqrt(var_arr[mo_idx] + eps);
}

// void InstanceNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
//                               NDArray& grad_arr, const NDArray& mean_arr,
//                               const NDArray& var_arr, float eps,
//                               const Stream& stream) {
//   HT_ASSERT_CUDA_DEVICE(out_grads);
//   HT_ASSERT_SAME_DEVICE(out_grads, in_arr); 
//   HT_ASSERT_SAME_DEVICE(out_grads, grad_arr); 
//   HT_ASSERT_SAME_DEVICE(out_grads, mean_arr);   
//   HT_ASSERT_SAME_DEVICE(out_grads, var_arr); 

//   CUDAStream hip_stream(stream);
//   hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
//   hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

//   int ndim = out_grads->ndim();
//   HT_ASSERT(ndim == 4);
//   size_t total_elements = 1;

//   hipdnnDataType_t datatype;
//   hipdnnIndicesType_t indicetype;
//   if (in_arr->dtype() == DataType::FLOAT32) {
//     datatype = HIPDNN_DATA_FLOAT;
//     indicetype = HIPDNN_32BIT_INDICES;
//   } else if (in_arr->dtype() == DataType::FLOAT64) {
//     datatype = HIPDNN_DATA_DOUBLE;
//     indicetype = HIPDNN_64BIT_INDICES;
//   } else if (in_arr->dtype() == DataType::FLOAT16) {
//     datatype = HIPDNN_DATA_HALF;
//     indicetype = HIPDNN_32BIT_INDICES;
//   }
//   #if defined(CUDNN_VERSION) && CUDNN_VERSION >= 8200
//   else if (in_arr->dtype() == DataType::BFLOAT16) {
//     datatype = CUDNN_DATA_BFLOAT16;
//     indicetype = HIPDNN_32BIT_INDICES;
//   }
//   #endif
//   else {
//     HT_LOG_INFO << "UNSUPPORTED TYPE:" << in_arr->dtype();
//   }

//   HT_ASSERT(ndim == 4);
//   int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);
//   size_t cpu_mem = ndim * sizeof(int);
//   int* dimA = (int*) malloc(cpu_mem);
//   int* strideA = (int*) malloc(cpu_mem);
//   int* dimC = (int*) malloc(cpu_mem);
//   int* strideC = (int*) malloc(cpu_mem);

//   int temp_strideA = 1;
//   int temp_strideC = 1;

//   for (int i = ndim - 1; i >= 0; --i) {
//     dimA[i] = (int) in_arr->shape(i);
//     dimC[i] = i < in_arr->ndim() - 2 ? (int) in_arr->shape(i) : 1;
//     strideA[i] = temp_strideA;
//     strideC[i] = temp_strideC;
//     temp_strideA *= dimA[i];
//     temp_strideC *= dimC[i];
//   }

//   for (int i = 0; i < ndim; ++i)
//     total_elements *= out_grads->shape(i);
//   int last2dim = out_grads->shape(ndim - 1) * out_grads->shape(ndim - 2);

//   size_t size = total_elements;
//   if (size == 0)
//     return;
//   dim3 blocks, threads;
//   threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
//   blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
//   HT_DISPATCH_FLOATING_TYPES(
//     in_arr->dtype(), spec_t, "CauculateGradCuda", [&]() {
//       #if defined(CUDNN_VERSION) && CUDNN_VERSION < 8200
//       if (in_arr->dtype() == DataType::BFLOAT16)
//         return;
//       #endif
//       spec_t* dscale = NULL;
//       DataPtr dscale_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideC * sizeof(spec_t));
//       dscale = (spec_t*) dscale_ptr.ptr;

//       spec_t* dbias = NULL;
//       DataPtr dbias_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideC * sizeof(spec_t));
//       dbias = (spec_t*) dbias_ptr.ptr;

//       spec_t* dy_mul_x = NULL;
//       DataPtr dy_mul_x_ptr = AllocFromMemoryPool(in_arr->device(), temp_strideA * sizeof(spec_t));
//       dy_mul_x = (spec_t*) dy_mul_x_ptr.ptr;

//       void* workspace = NULL;
//       DataPtr workspace_ptr = in_arr->dtype() == DataType::FLOAT16 ? AllocFromMemoryPool(in_arr->device(), temp_strideA * sizeof(float))
//                                                                    : AllocFromMemoryPool(in_arr->device(), temp_strideA * sizeof(spec_t));
//       workspace = (void*) workspace_ptr.ptr;

//       spec_t one = 1.0f;
//       spec_t zero = 0.0f;

//       float one_f = 1.0f;
//       float zero_f = 0.0f;

//       hipdnnReduceTensorDescriptor_t rtd;
//       CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
//       CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
//         rtd, HIPDNN_REDUCE_TENSOR_ADD, in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16 ? HIPDNN_DATA_FLOAT : datatype, HIPDNN_PROPAGATE_NAN,
//         HIPDNN_REDUCE_TENSOR_NO_INDICES, indicetype));

//       hipdnnTensorDescriptor_t adesc;
//       hipdnnTensorDescriptor_t cdesc;
//       CUDNN_CALL(hipdnnCreateTensorDescriptor(&adesc));
//       CUDNN_CALL(hipdnnCreateTensorDescriptor(&cdesc));

//       CUDNN_CALL(
//         hipdnnSetTensorNdDescriptor(adesc, datatype, ndim, dimA, strideA));
//       CUDNN_CALL(
//         hipdnnSetTensorNdDescriptor(cdesc, datatype, ndim, dimC, strideC));

//       if (in_arr->dtype() == DataType::FLOAT16) {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(float), &one_f,
//           adesc, (const void*) out_grads->data_ptr<void>(), &zero_f, cdesc,
//           (void*) dbias));     
//       }
//       else {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
//           adesc, (const void*) out_grads->data_ptr<void>(), &zero, cdesc,
//           (void*) dbias));      
//       }

//       auto op = kmultiplies<spec_t, spec_t>();

//       binary_elewise_kernel<spec_t, spec_t><<<blocks, threads, 0, hip_stream>>>(
//         out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
//         size, op, dy_mul_x);
//       if (in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16) {  
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(float), &one_f,
//           adesc, (const void*) dy_mul_x, &zero_f, cdesc,
//           (void*) dscale));  
//       }
//       else {
//         CUDNN_CALL(hipdnnReduceTensor(
//           handle, rtd, NULL, 0, (void*) workspace, temp_strideA * sizeof(spec_t), &one,
//           adesc, (const void*) dy_mul_x, &zero, cdesc,
//           (void*) dscale));  
//       }
//       calculate_grad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
//         out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
//         mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
//         dscale, dbias,
//         grad_arr->data_ptr<spec_t>(), last2dim, eps, size);
//       FreeToMemoryPool(dscale_ptr);
//       FreeToMemoryPool(dbias_ptr);
//       FreeToMemoryPool(dy_mul_x_ptr);
//       FreeToMemoryPool(workspace_ptr);
//     });
//   free(dimA);
//   free(strideA);
//   free(dimC);
//   free(strideC);
// }

void InstanceNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
                              NDArray& grad_arr, const NDArray& mean_arr,
                              const NDArray& var_arr, float eps,
                              const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(out_grads);
  HT_ASSERT_SAME_DEVICE(out_grads, in_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, grad_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, mean_arr);   
  HT_ASSERT_SAME_DEVICE(out_grads, var_arr); 

  int ndim = out_grads->ndim();
  HT_ASSERT(ndim == 4);
  size_t total_elements = 1;


  HT_ASSERT(ndim == 4);
  int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);

  for (int i = 0; i < ndim; ++i)
    total_elements *= out_grads->shape(i);
  int last2dim = out_grads->shape(ndim - 1) * out_grads->shape(ndim - 2);

  size_t size = total_elements;
  if (size == 0)
    return;
  
  auto device_id = out_grads->device().index();
  hetu::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  NDArray dbias_arr = NDArray::sum(out_grads, {2, 3}, true, stream.stream_index());
  NDArray dy_mul_x_arr = NDArray::mul(out_grads, in_arr, stream.stream_index());
  NDArray dscale_arr = NDArray::sum(dy_mul_x_arr, {2, 3}, true, stream.stream_index());
  HT_DISPATCH_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "CauculateGradCuda", [&]() {
      calculate_grad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
        dscale_arr->data_ptr<spec_t>(), dbias_arr->data_ptr<spec_t>(),
        grad_arr->data_ptr<spec_t>(), last2dim, eps, size);
    });
  NDArray::MarkUsedBy({out_grads, in_arr, grad_arr, mean_arr, var_arr,
                       dbias_arr, dy_mul_x_arr, dscale_arr},
                      stream);
}

} // namespace impl
} // namespace hetu
