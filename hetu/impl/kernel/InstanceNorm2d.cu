#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/cuda/CUDADnn.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void minus_mean_n_square_kernel1(const spec_t* in_arr,
                                            const spec_t* mean, spec_t* out_arr,
                                            int last_2dim, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  spec_t temp = in_arr[idx] - mean[idx / last_2dim];
  out_arr[idx] = temp * temp;
}

template <typename spec_t>
__global__ void std_normal_transform(const spec_t* in_arr,
                                     const spec_t* mean_arr,
                                     const spec_t* var_arr, spec_t* out_arr,
                                     int last_2dim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / last_2dim;
  out_arr[idx] =
    (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
}

void InstanceNormCuda(const NDArray& in_arr, NDArray& mean_arr,
                      NDArray& var_arr, NDArray& out_arr, float eps,
                      const Stream& stream) {
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype;
  hipdnnIndicesType_t indicetype;
  if (in_arr->dtype() == DataType::FLOAT32) {
    datatype = HIPDNN_DATA_FLOAT;
    indicetype = HIPDNN_32BIT_INDICES;
  } else if (in_arr->dtype() == DataType::FLOAT64) {
    datatype = HIPDNN_DATA_DOUBLE;
    indicetype = HIPDNN_64BIT_INDICES;
  }

  int ndim = in_arr->ndim();
  HT_ASSERT(ndim == 4);
  int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);
  size_t cpu_mem = ndim * sizeof(int);
  int* dimA = (int*) malloc(cpu_mem);
  int* strideA = (int*) malloc(cpu_mem);
  int* dimC = (int*) malloc(cpu_mem);
  int* strideC = (int*) malloc(cpu_mem);

  int temp_strideA = 1;
  int temp_strideC = 1;

  for (int i = ndim - 1; i >= 0; --i) {
    dimA[i] = (int) in_arr->shape(i);
    dimC[i] = i < in_arr->ndim() - 2 ? (int) in_arr->shape(i) : 1;
    strideA[i] = temp_strideA;
    strideC[i] = temp_strideC;
    temp_strideA *= dimA[i];
    temp_strideC *= dimC[i];
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "InstanceNormCuda", [&]() {
      size_t size = temp_strideA * sizeof(spec_t);

      float one = 1.0f;
      float zero = 0.0f;

      hipdnnReduceTensorDescriptor_t rtd;
      CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
      CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
        rtd, HIPDNN_REDUCE_TENSOR_AVG, datatype, HIPDNN_PROPAGATE_NAN,
        HIPDNN_REDUCE_TENSOR_NO_INDICES, indicetype));

      hipdnnTensorDescriptor_t adesc;
      hipdnnTensorDescriptor_t cdesc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&adesc));
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&cdesc));

      CUDNN_CALL(
        hipdnnSetTensorNdDescriptor(adesc, datatype, ndim, dimA, strideA));
      CUDNN_CALL(
        hipdnnSetTensorNdDescriptor(cdesc, datatype, ndim, dimC, strideC));

      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, (void*) out_arr->data_ptr<void>(), size, &one,
        adesc, (const void*) in_arr->data_ptr<void>(), &zero, cdesc,
        (void*) mean_arr->data_ptr<void>()));
      dim3 blocks, threads;
      threads.x = MIN(temp_strideA, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
      blocks.x = DIVUP(temp_strideA, HT_DEFAULT_NUM_THREADS_PER_BLOCK);

      minus_mean_n_square_kernel1<spec_t><<<blocks, threads, 0, hip_stream>>>(
        in_arr->data_ptr<spec_t>(), mean_arr->data_ptr<spec_t>(),
        out_arr->data_ptr<spec_t>(), last_2dim, temp_strideA);

      CUDNN_CALL(hipdnnReduceTensor(
        handle, rtd, NULL, 0, (void*) out_arr->data_ptr<void>(), size, &one,
        adesc, (const void*) out_arr->data_ptr<void>(), &zero, cdesc,
        (void*) var_arr->data_ptr<void>()));

      threads.x = MIN(temp_strideA, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
      blocks.x = DIVUP(temp_strideA, HT_DEFAULT_NUM_THREADS_PER_BLOCK);

      std_normal_transform<spec_t><<<blocks, threads, 0, hip_stream>>>(
        in_arr->data_ptr<spec_t>(), mean_arr->data_ptr<spec_t>(),
        var_arr->data_ptr<spec_t>(), out_arr->data_ptr<spec_t>(), last_2dim,
        eps, temp_strideA);

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(adesc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(cdesc));
      CUDNN_CALL(hipdnnDestroyReduceTensorDescriptor(rtd));
    });
  free(dimA);
  free(dimC);
  free(strideA);
  free(strideC);
  return;
}

template <typename spec_t>
__global__ void calculate_grad_kernel(const spec_t* out_grads,
                                      const spec_t* in_arr,
                                      const spec_t* mean_arr,
                                      const spec_t* var_arr, spec_t* grad_arr,
                                      size_t last2dim, float eps, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t mo_idx = idx / last2dim;
  float y = (in_arr[idx] - mean_arr[mo_idx]) / sqrtf(var_arr[mo_idx] + eps);
  grad_arr[idx] = out_grads[idx] * (1.0 - 1.0 / (float) last2dim - y * y) /
    sqrtf(var_arr[mo_idx] + eps);
}

void InstanceNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
                              NDArray& grad_arr, const NDArray& mean_arr,
                              const NDArray& var_arr, float eps,
                              const Stream& stream) {
  /*
    already have mean and var, we directly get y = x-u / sigma
    the grad_arr = out_grad * (1 - 1/WH - y^2) / sigma
  */
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hetu::impl::GetCudnnHandle(hip_stream.device_id());

  int ndim = out_grads->ndim();
  HT_ASSERT(ndim == 4);
  size_t total_elements = 1;
  for (int i = 0; i < ndim; ++i)
    total_elements *= out_grads->shape(i);
  int last2dim = out_grads->shape(ndim - 1) * out_grads->shape(ndim - 2);

  size_t size = total_elements;
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "CauculateGradCuda", [&]() {
      calculate_grad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        out_grads->data_ptr<spec_t>(), in_arr->data_ptr<spec_t>(),
        mean_arr->data_ptr<spec_t>(), var_arr->data_ptr<spec_t>(),
        grad_arr->data_ptr<spec_t>(), last2dim, eps, size);
    });
}

} // namespace impl
} // namespace hetu
