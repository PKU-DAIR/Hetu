#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/offset_calculator.cuh"
#include "hetu/impl/kernel/Vectorized.cuh"
#include "hetu/impl/cuda/CUB.h"



namespace hetu{
namespace impl{


void MaskedScatterCuda(const NDArray& input, const NDArray& mask, const NDArray& source,
                NDArray& output, const Stream& stream) {
  hetu::cuda::CUDADeviceGuard guard(stream.device_index());
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, mask);
  HT_ASSERT_SAME_DEVICE(input, source);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, mask);
  HT_ASSERT_SAME_SHAPE(input, output);
  auto maskPrefixSum = NDArray::empty(input->shape(), input->device(), kInt64, stream.stream_index());
  auto maskPrefixSum_data = maskPrefixSum->data_ptr<int64_t>();
  auto mask_data = mask->data_ptr<int64_t>();
  exclusive_scan(mask_data, maskPrefixSum_data, hipcub::Sum(), (int64_t)0, input->numel(), stream); 
  size_t size = input->numel();

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "MaskedScatterCuda", [&]() {
      auto source_ptr = source->data_ptr<spec_t>();
      using InType = std::tuple<spec_t, int64_t, int64_t>;
      using OutType = thrust::tuple<spec_t>;
      launch_loop_kernel<InType, OutType>({input, mask, maskPrefixSum}, {output}, size, stream,
                                                  [source_ptr] __device__ (spec_t in, int64_t mask, int64_t maskPrefixSum) -> spec_t {
                                                    if(mask) return static_cast<spec_t>(source_ptr[maskPrefixSum]);
                                                    else return in;
                                                 });
  });

  NDArray::MarkUsedBy({input, mask, source, output, maskPrefixSum}, stream);
}

}
}