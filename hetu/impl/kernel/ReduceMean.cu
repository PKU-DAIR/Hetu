#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/kernel/Reduce.cuh"

namespace hetu {
namespace impl {

template <typename spec_t, typename arg_t>
struct MeanOps {
  arg_t factor;
  MeanOps(arg_t factor) : factor(factor) {}

  inline __device__ spec_t project(arg_t val) const {
    return static_cast<spec_t>(val * factor);
  }

  __device__ arg_t reduce(arg_t acc, arg_t val) const {
    return acc + val;
  }
};

void ReduceMeanCuda(const NDArray& in_arr, NDArray& out_arr, const int64_t* axes,
                   int64_t num_ax, const Stream& stream) {
  size_t reduce_num = 1;
  for (int64_t i = 0; i < num_ax; i++) {
    reduce_num *= in_arr->shape(axes[i]);
  }
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "ReduceMeanCuda", [&]() {
        using arg_t = opmath_type<spec_t>;
        using MeanFunctor = MeanOps<spec_t, arg_t>;
        arg_t factor = static_cast<arg_t>(1.0 / reduce_num);
        // TODO: Optimize it. Small size memcpy is inefficient.
        MeanFunctor ops{factor};
        CUDAStream hip_stream(stream);
        DataPtr ops_cu_ptr = AllocFromMemoryPool(in_arr->device(), sizeof(MeanFunctor));
        MeanFunctor* ops_cu = reinterpret_cast<MeanFunctor*>(ops_cu_ptr.ptr);
        CudaMemcpyAsync(ops_cu, &ops, sizeof(MeanFunctor), hipMemcpyHostToDevice, hip_stream);

        launch_reduce_kernel<spec_t, arg_t>(in_arr, out_arr, axes, num_ax,
                                            *ops_cu, 0., stream);
        FreeToMemoryPool(ops_cu_ptr);
    });
}

} // namespace impl
} // namespace hetu
