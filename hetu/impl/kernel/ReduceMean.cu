#include "hetu/core/ndarray.h"
#include "hetu/core/memory_pool.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/kernel/Reduce.cuh"

namespace hetu {
namespace impl {

template <typename acc_t>
struct MeanOp {
  acc_t factor;
  MeanOp(acc_t factor) : factor(factor) {}

  inline __device__ acc_t project(acc_t val) const {
    return val * factor;
  }

  __device__ acc_t reduce(acc_t acc, acc_t val) const {
    return acc + val;
  }
};

template <typename spec_t, typename acc_t = spec_t, typename out_t = spec_t>
static void mean_functor(const NDArray& in_arr, NDArray& out_arr, const int64_t* axes,
                         int64_t num_ax, size_t reduce_num, const Stream& stream) {
  using MeanOp_t = MeanOp<acc_t>;
  acc_t factor = static_cast<acc_t>(1.0 / reduce_num);
  MeanOp_t ops{factor};
  auto device_id = in_arr->device().index();
  hetu::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  auto ops_arr = 
    hetu::cuda::to_byte_ndarray(reinterpret_cast<uint8_t*>(&ops),
                                sizeof(MeanOp_t), device_id);
  launch_reduce_kernel<spec_t, out_t, acc_t>(in_arr, out_arr, axes, num_ax,
                                             *(ops_arr->data_ptr<MeanOp_t>()), 0., stream);
  NDArray::MarkUsedBy({ops_arr}, stream);
}

void ReduceMeanCuda(const NDArray& in_arr, NDArray& out_arr, const int64_t* axes,
                   int64_t num_ax, const Stream& stream) {
  size_t reduce_num = 1;
  for (int64_t i = 0; i < num_ax; i++) {
    reduce_num *= in_arr->shape(axes[i]);
  }
  if (out_arr->dtype() == DataType::FLOAT16) {
    mean_functor<hetu::float16, float>(in_arr, out_arr, axes, num_ax, reduce_num, stream);
  } else if (in_arr->dtype() == DataType::FLOAT16 && out_arr->dtype() == DataType::FLOAT32) {
    mean_functor<hetu::float16, float, float>(in_arr, out_arr, axes, num_ax, reduce_num, stream);
  } else if (out_arr->dtype() == DataType::BFLOAT16) {
    mean_functor<hetu::bfloat16, float>(in_arr, out_arr, axes, num_ax, reduce_num, stream);
  } else if (in_arr->dtype() == DataType::BFLOAT16 && out_arr->dtype() == DataType::FLOAT32) {
    mean_functor<hetu::bfloat16, float, float>(in_arr, out_arr, axes, num_ax, reduce_num, stream);
  } else {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    out_arr->dtype(), spec_t, "ReduceMeanCuda", [&]() {
      mean_functor<spec_t>(in_arr, out_arr, axes, num_ax, reduce_num, stream);
    });
  }
}

} // namespace impl
} // namespace hetu
