#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"
#include <chrono>

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void embedding_lookup_kernel(const spec_t* input, const int64_t* ids,
                                        size_t size, size_t length,
                                        size_t input_row, spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int64_t id = ids[idx / length]; 
  if (id < 0 || id >= input_row) {
    output[idx] = 0;
  } else {
    output[idx] = input[length * id + (idx % length)];
  }
}

template <typename spec_t>
__global__ void array_zero_set_kernel(spec_t* input, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  input[idx] = 0;
}

template <typename spec_t>
__global__ void embedding_lookup_gradient_kernel(const spec_t* output_grad,
                                                 const int64_t* ids, size_t size,
                                                 size_t length,
                                                 spec_t* input_grad) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int id = int(ids[idx / length]);
  hetu::cuda::AtomicAdd((input_grad + length * id + (idx % length)), (output_grad[idx]));
}

void EmbeddingLookupCuda(const NDArray& input, const NDArray& id,
                         NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, id);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT(input->ndim() == 2)
    << "input_dim is invalid.Expect 2,but get " << input->ndim();

  for (int i = 0; i < output->ndim(); i++) {
    if (i < output->ndim() - 1) {
      HT_ASSERT(id->shape(i) == output->shape(i));
    } else if (i == output->ndim() - 1) {
      HT_ASSERT(input->shape(1) == output->shape(i));
    }
  }
  size_t input_row = input->shape(0);
  size_t length = input->shape(1);
  size_t size = id->numel() * input->shape(1);
  if (size == 0 || input_row == 0 || length == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "EmbbedingLookupCuda", [&]() {
      embedding_lookup_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), id->data_ptr<int64_t>(), size, length,
        input_row, output->data_ptr<spec_t>());
    });
}

void EmbeddingLookupGradientCuda(const NDArray& output_grad, const NDArray& id,
                                 NDArray& input_grad, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(output_grad);
  HT_ASSERT_SAME_DEVICE(output_grad, id);
  HT_ASSERT_SAME_DEVICE(output_grad, input_grad);
  HT_ASSERT(input_grad->ndim() == 2)
    << "input_dim is invalid.Expect 2,but get " << input_grad->ndim();

  for (int i = 0; i < output_grad->ndim(); i++) {
    if (i < output_grad->ndim() - 1) {
      HT_ASSERT(id->shape(i) == output_grad->shape(i));
    } else if (i == output_grad->ndim() - 1) {
      HT_ASSERT(input_grad->shape(1) == output_grad->shape(i));
    }
  }
  size_t length = input_grad->shape(1);
  size_t size = input_grad->numel();
  if (size == 0 || length == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_FLOATING_TYPES(
    input_grad->dtype(), spec_t, "ArrayZeroSet", [&]() {
      array_zero_set_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input_grad->data_ptr<spec_t>(), size);
    });
  size_t size2 = output_grad->numel();
  threads.x = MIN(size2, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size2, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_FLOATING_TYPES(
    input_grad->dtype(), spec_t, "EmbeddingLookupGradientCuda", [&]() {
      embedding_lookup_gradient_kernel<spec_t>
        <<<blocks, threads, 0, hip_stream>>>(
          output_grad->data_ptr<spec_t>(), id->data_ptr<int64_t>(), size2, length,
          input_grad->data_ptr<spec_t>());
    });
}

} // namespace impl
} // namespace hetu
