#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void add_const_kernel(const spec_t* input, spec_t value, size_t size,
                                 spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    output[idx] = input[idx] + value;
}

void AddConstCuda(const NDArray& input, double value, NDArray& output,
                  const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "AddConstCuda", [&]() {
      add_const_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), static_cast<spec_t>(value), size,
        output->data_ptr<spec_t>());
    });
}

} // namespace impl
} // namespace hetu
