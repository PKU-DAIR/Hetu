#include "hetu/core/ndarray.h"
#include "hetu/core/stream.h"
#include "hetu/impl/cuda/CUDABlas.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

void MatVecMulCuda(const NDArray& a, bool trans, const NDArray& x,
                NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(a);
  HT_ASSERT_SAME_DEVICE(a, x);
  HT_ASSERT_SAME_DEVICE(a, output);
  HT_ASSERT_NDIM(a, 2);
  HT_ASSERT_NDIM(x, 1);
  HT_ASSERT_NDIM(output, 1);
  HT_ASSERT_SAME_DTYPE(a, x);
  HT_ASSERT_SAME_DTYPE(a, output);

  hipblasHandle_t cublas_handle = GetCublasHandle(output->device().index());
  hetu::cuda::CUDADeviceGuard guard(output->device().index());
  int32_t m = a->shape(1);
  int32_t n = a->shape(0);

  HT_DISPATCH_FLOATING_TYPES(output->dtype(), spec_t, "MatVecMul", [&]() {
    spec_t alpha = 1, beta = 0;
    float alpha_f = 1, beta_f = 0;
    if (output->dtype() == DataType::FLOAT16 || output->dtype() == DataType::BFLOAT16) {
      cublas_gemv<spec_t>(cublas_handle, !trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                          m, n, static_cast<const void*>(&alpha_f),
                          a->data_ptr<spec_t>(), m,
                          x->data_ptr<spec_t>(), 1, static_cast<const void*>(&beta_f),
                          output->data_ptr<spec_t>(), 1);
    }
    else {
      cublas_gemv<spec_t>(cublas_handle, !trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                          m, n, static_cast<const void*>(&alpha),
                          a->data_ptr<spec_t>(), m,
                          x->data_ptr<spec_t>(), 1, static_cast<const void*>(&beta),
                          output->data_ptr<spec_t>(), 1);
    }
  });
  NDArray::MarkUsedBy({a, x, output}, stream);
}

} // namespace impl
} // namespace hetu
