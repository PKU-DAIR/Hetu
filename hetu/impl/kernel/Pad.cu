#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void
pad_kernel(const spec_t* input_data, spec_t* output_data, size_t begin_N,
           size_t end_N, size_t N, size_t begin_C, size_t end_C, size_t C,
           size_t begin_H, size_t end_H, size_t H, size_t begin_W, size_t end_W,
           size_t W, spec_t constant_value) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N * C * H * W)
    return;
  size_t idx_N = idx / (C * H * W);
  size_t idx_C = idx % (C * H * W) / (H * W);
  size_t idx_H = idx % (H * W) / W;
  size_t idx_W = idx % W;
  if (idx_N >= begin_N && idx_N < end_N && idx_C >= begin_C && idx_C < end_C &&
      idx_H >= begin_H && idx_H < end_H && idx_W >= begin_W && idx_W < end_W) {
    output_data[idx] =
      input_data[(((idx_N - begin_N) * (end_C - begin_C) + idx_C - begin_C) *
                    (end_H - begin_H) +
                  idx_H - begin_H) *
                   (end_W - begin_W) +
                 idx_W - begin_W];
  } else {
    output_data[idx] = constant_value;
  }
}

template <typename spec_t>
__global__ void
pad_gradient_kernel(const spec_t* output_grad, spec_t* input_grad, size_t N,
                    size_t C, size_t H, size_t W, size_t begin_N,
                    size_t begin_C, size_t begin_H, size_t begin_W,
                    size_t out_N, size_t out_C, size_t out_H, size_t out_W) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N * C * H * W)
    return;
  size_t idx_N = idx / (C * H * W);
  size_t idx_C = idx % (C * H * W) / (H * W);
  size_t idx_H = idx % (H * W) / W;
  size_t idx_W = idx % W;
  input_grad[idx] = output_grad[(
    (((idx_N + begin_N) * out_C + idx_C + begin_C) * out_H + idx_H + begin_H) *
      out_W +
    idx_W + begin_W)];
}

void PadCuda(const NDArray& input, NDArray& output, const HTShape& paddings,
             const Stream& stream, std::string mode = "constant",
             double constant_values = 0) {
  HT_ASSERT(input->is_cuda()) << "Input is not on a host device.";
  HT_ASSERT(output->is_cuda()) << "Output is not on a host device.";
  HT_ASSERT(input->device() == output->device())
    << "Input and output are not on the same host device. "
    << "Devices: (input) " << input->device() << " vs. (output) "
    << output->device();
  size_t pad_len = paddings.size();
  size_t len = pad_len;
  size_t endpoint[8];
  for (int i = 0; i < 4; i++) {
    if (i < (4 - len / 2)) {
      HT_ASSERT((input->shape(i)) == (output->shape(i)));
      // endpoint[i * 2] = input->shape(i) - 1;
      // endpoint[i * 2 + 1] = endpoint[i * 2] + 1;
      endpoint[i * 2] = 0;
      endpoint[i * 2 + 1] = input->shape(i);
    } else {
      HT_ASSERT((input->shape(i) + paddings[(i - (4 - len / 2)) * 2] +
                 paddings[(i - (4 - len / 2)) * 2 + 1]) == (output->shape(i)));
      endpoint[i * 2] = paddings[(i - (4 - len / 2)) * 2];
      endpoint[i * 2 + 1] = paddings[(i - (4 - len / 2)) * 2] + input->shape(i);
    }
  }
  size_t size = output->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  if (mode == "constant") {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      input->dtype(), spec_t, "PadCuda", [&]() {
        pad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), endpoint[0],
          endpoint[1], output->shape(0), endpoint[2], endpoint[3],
          output->shape(1), endpoint[4], endpoint[5], output->shape(2),
          endpoint[6], endpoint[7], output->shape(3), constant_values);
      });
  }
}

void PadGradientCuda(const NDArray& output_grad, NDArray& input_grad,
                     const HTShape& paddings, const Stream& stream,
                     std::string mode = "constant") {
  HT_ASSERT(output_grad->is_cuda()) << "Output_grad is not on a host device.";
  HT_ASSERT(input_grad->is_cuda()) << "Input_grad is not on a host device.";
  HT_ASSERT(input_grad->device() == output_grad->device())
    << "input and output grads are not on the same host device. "
    << "Devices: (input_grad) " << input_grad->device() << " vs. (output_grad) "
    << output_grad->device();
  size_t pad_len = paddings.size();
  size_t len = pad_len;
  size_t begin_p[4];
  size_t N = input_grad->shape(0);
  size_t C = input_grad->shape(1);
  size_t H = input_grad->shape(2);
  size_t W = input_grad->shape(3);

  size_t out_N = output_grad->shape(0);
  size_t out_C = output_grad->shape(1);
  size_t out_H = output_grad->shape(2);
  size_t out_W = output_grad->shape(3);

  for (int i = 0; i < 4; i++) {
    if (i < (4 - len / 2)) {
      begin_p[i] = 0;
    } else {
      begin_p[i] = paddings[(i - (4 - len / 2)) * 2];
    }
  }
  size_t size = input_grad->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  if (mode == "constant") {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      input_grad->dtype(), spec_t, "PadGradientCuda", [&]() {
        pad_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          output_grad->data_ptr<spec_t>(), input_grad->data_ptr<spec_t>(), N, C,
          H, W, begin_p[0], begin_p[1], begin_p[2], begin_p[3], out_N, out_C,
          out_H, out_W);
      });
  }
}

} // namespace impl
} // namespace hetu
