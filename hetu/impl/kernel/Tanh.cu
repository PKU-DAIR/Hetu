#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/cuda_math.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void tanh_kernel(const spec_t* input, size_t size, spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = hetu::cuda::cuda_tanh(input[idx]);
}

template <typename spec_t>
__global__ void tanh_gradient_kernel(const spec_t* input,
                                     const spec_t* output_grad, size_t size,
                                     spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = (1 - input[idx] * input[idx]) * output_grad[idx];
}

void TanhCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_EXCHANGABLE(input, output);

  size_t size = output->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_FLOATING_TYPES(input->dtype(), spec_t, "TanhCuda", [&]() {
    tanh_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
      input->data_ptr<spec_t>(), size, output->data_ptr<spec_t>());
  });
  NDArray::MarkUsedBy({input, output}, stream);
}

void TanhGradientCuda(const NDArray& input, const NDArray& output_grad,
                      NDArray& input_grad, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output_grad);
  HT_ASSERT_SAME_DEVICE(input, input_grad);
  HT_ASSERT_EXCHANGABLE(input, output_grad);
  HT_ASSERT_EXCHANGABLE(input, input_grad);

  size_t size = input_grad->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "TanhGradientCuda", [&]() {
      tanh_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output_grad->data_ptr<spec_t>(), size,
        input_grad->data_ptr<spec_t>());
    });
  NDArray::MarkUsedBy({input, output_grad, input_grad}, stream);
}

} // namespace impl
} // namespace hetu
