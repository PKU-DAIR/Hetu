#include "hip/hip_runtime.h"
#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"

namespace hetu {
namespace impl {

template <typename spec_t>
__global__ void reciprocal_kernel(const spec_t* input, size_t size,
                                  spec_t* output) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = static_cast<spec_t>(1) / input[idx];
}

void ReciprocalCuda(const NDArray& input, NDArray& output,
                    const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_CUDA_DEVICE(output);
  HT_ASSERT_EXCHANGABLE(input, output);
  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hetu::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  HT_DISPATCH_FLOATING_TYPES(input->dtype(), spec_t, "ReciprocalCuda", [&]() {
    reciprocal_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
      input->data_ptr<spec_t>(), size, output->data_ptr<spec_t>());
  });
}

} // namespace impl
} // namespace hetu
