#include "hetu/core/ndarray.h"
#include "hetu/impl/stream/CUDAStream.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/utils/cuda_utils.h"
#include "hetu/impl/utils/offset_calculator.cuh"
#include "hetu/impl/kernel/Vectorized.cuh"

namespace hetu {
namespace impl {

void OppositeCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  bool contiguous = input->is_contiguous() && output->is_contiguous();
  if (contiguous) {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      input->dtype(), spec_t, "OppositeCuda", [&]() {
        launch_vectorized_unary_kernel(input->data_ptr<spec_t>(), size,
                                       output->data_ptr<spec_t>(), stream,
                                       [=] __device__ (spec_t x) -> spec_t {
                                         return -x;
                                       });
    });
  } else {
    constexpr int unroll_factor = sizeof(DataType2Size(output->dtype())) >= 4 ? 2 : 4;
    dim3 block(128);
    dim3 grid(DIVUP(size, unroll_factor * block.x));
    NDArray in_offset_calculator_arr, out_offset_calculator_arr;
    OffsetCalculator *in_offset_calculator, *out_offset_calculator;
    std::tie(in_offset_calculator_arr, in_offset_calculator) =
      AllocOffsetCalculator(input, stream);
    std::tie(out_offset_calculator_arr, out_offset_calculator) = 
      AllocOffsetCalculator(output, stream);
    CUDAStream hip_stream(stream);
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      input->dtype(), spec_t, "OppositeCuda", [&]() {
        unary_kernel<128, unroll_factor><<<grid, block, 0, hip_stream>>>(
          input->data_ptr<spec_t>(), size, output->data_ptr<spec_t>(),
          [=] __device__ (spec_t x) -> spec_t {
            return -x;
          }, in_offset_calculator, out_offset_calculator);
    });
    NDArray::MarkUsedBy({in_offset_calculator_arr, out_offset_calculator_arr}, stream);
  }
  NDArray::MarkUsedBy({input, output}, stream);
}

} // namespace impl
} // namespace hetu
