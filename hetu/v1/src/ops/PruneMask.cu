#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void less_const_kernel(const float *input, float *output,
                                  float threshold, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (abs(input[ind]) < threshold);
}

__global__ void less_const_kernel_1d_buffer(const float *input, float *output,
                                            float threshold, size_t dim,
                                            size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    const float *src_ptr = input + ind * dim;
    float result = 0;
    for (size_t i = 0; i < dim; ++i) {
        result += (abs(src_ptr[i]) < threshold);
    }
    output[ind] = result;
}

int DLGpuNumLessThan(const DLArrayHandle input, DLArrayHandle middle,
                     DLArrayHandle output, float threshold, int *axes,
                     int num_ax, DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(input);
    size_t dim = input->shape[1];
    const float *input_data = (const float *)input->data;
    float *middle_data = (float *)middle->data;
    if (middle->ndim == 1) {
        dim = size / middle->shape[0];
        size = middle->shape[0];
    }
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (middle->ndim == 1) {
        if (stream_handle)
            less_const_kernel_1d_buffer<<<
                blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                input_data, middle_data, threshold, dim, size);
        else
            less_const_kernel_1d_buffer<<<blocks, threads>>>(
                input_data, middle_data, threshold, dim, size);
    } else {
        if (stream_handle)
            less_const_kernel<<<blocks, threads, 0,
                                *(hipStream_t *)stream_handle->handle>>>(
                input_data, middle_data, threshold, size);
        else
            less_const_kernel<<<blocks, threads>>>(input_data, middle_data,
                                                   threshold, size);
    }
    return DLGpuReduceSum(middle, output, axes, num_ax, stream_handle);
}

__global__ void set_less_const_kernel(float *arr, float threshold,
                                      size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    if (abs(arr[ind]) < threshold) {
        arr[ind] = 0;
    }
}

int DLGpuSetLessThan(const DLArrayHandle arr, float threshold,
                     DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(arr);
    float *arr_data = (float *)arr->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (stream_handle)
        set_less_const_kernel<<<blocks, threads, 0,
                                *(hipStream_t *)stream_handle->handle>>>(
            arr_data, threshold, size);
    else
        set_less_const_kernel<<<blocks, threads>>>(arr_data, threshold, size);
    return 0;
}

__global__ void set_mask_less_const_kernel(float *arr, int *mask,
                                           float threshold, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    if (abs(arr[ind]) < threshold) {
        arr[ind] = 0.;
        mask[ind] = 0;
    } else {
        mask[ind] = 1;
    }
}

int DLGpuSetMaskLessThan(DLArrayHandle arr, DLArrayHandle mask, float threshold,
                         DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(arr);
    float *arr_data = (float *)arr->data;
    int *mask_data = (int *)mask->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (stream_handle)
        set_mask_less_const_kernel<<<blocks, threads, 0,
                                     *(hipStream_t *)stream_handle->handle>>>(
            arr_data, mask_data, threshold, size);
    else
        set_mask_less_const_kernel<<<blocks, threads>>>(arr_data, mask_data,
                                                        threshold, size);
    return 0;
}

__global__ void get_larger_than_kernel_feature_dimension(const float *arr,
                                                         const float *threshold,
                                                         int *mask,
                                                         size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    mask[ind] = (abs(arr[ind]) >= threshold[ind]);
}

__global__ void get_larger_than_kernel_feature(const float *arr,
                                               const float *threshold,
                                               int *mask, size_t size,
                                               size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    size_t rind = ind / dim;
    mask[ind] = (abs(arr[ind]) >= threshold[rind]);
}

__global__ void get_larger_than_kernel_dimension(const float *arr,
                                                 const float *threshold,
                                                 int *mask, size_t size,
                                                 size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    size_t cind = ind % dim;
    mask[ind] = (abs(arr[ind]) >= threshold[cind]);
}

__global__ void get_larger_than_kernel_global(const float *arr,
                                              const float *threshold, int *mask,
                                              size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    mask[ind] = (abs(arr[ind]) >= threshold[0]);
}

int DLGpuGetLargerThan(const DLArrayHandle input, const DLArrayHandle threshold,
                       DLArrayHandle mask,
                       DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = ArrSize(input);
    size_t dim = input->shape[1];
    const float *input_data = (const float *)input->data;
    const float *thres_data = (const float *)threshold->data;
    int *mask_data = (int *)mask->data;
    size_t thres_ndim = threshold->ndim;
    size_t last_dim = threshold->shape[thres_ndim - 1];
    bool use_feature = (thres_ndim > 1);
    bool use_dimension = (last_dim > 1);
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    assert(stream_handle != NULL);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    if (use_feature && use_dimension) {
        get_larger_than_kernel_feature_dimension<<<blocks, threads, 0,
                                                   stream>>>(
            input_data, thres_data, mask_data, size);
    } else if (!use_feature && use_dimension) {
        get_larger_than_kernel_dimension<<<blocks, threads, 0, stream>>>(
            input_data, thres_data, mask_data, size, dim);
    } else if (use_feature && !use_dimension) {
        get_larger_than_kernel_feature<<<blocks, threads, 0, stream>>>(
            input_data, thres_data, mask_data, size, dim);
    } else {
        get_larger_than_kernel_global<<<blocks, threads, 0, stream>>>(
            input_data, thres_data, mask_data, size);
    }
    return 0;
}

__global__ void less_tensor_kernel_feature_dimension(const float *input,
                                                     float *output,
                                                     const float *threshold,
                                                     size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (abs(input[ind]) < threshold[ind]);
}

__global__ void less_tensor_kernel_feature(const float *input, float *output,
                                           const float *threshold, size_t size,
                                           size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (abs(input[ind]) < threshold[ind / dim]);
}

__global__ void less_tensor_kernel_dimension(const float *input, float *output,
                                             const float *threshold,
                                             size_t size, size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (abs(input[ind]) < threshold[ind % dim]);
}

__global__ void less_tensor_kernel_global(const float *input, float *output,
                                          const float *threshold, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (abs(input[ind]) < threshold[0]);
}

int DLGpuNumLessThanTensorThreshold(const DLArrayHandle input,
                                    DLArrayHandle middle, DLArrayHandle output,
                                    const DLArrayHandle threshold, int *axes,
                                    int num_ax,
                                    DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = ArrSize(input);
    size_t dim = input->shape[1];
    const float *input_data = (const float *)input->data;
    float *middle_data = (float *)middle->data;
    const float *thres_data = (const float *)threshold->data;
    size_t thres_ndim = threshold->ndim;
    size_t last_dim = threshold->shape[thres_ndim - 1];
    bool use_feature = (thres_ndim > 1);
    bool use_dimension = (last_dim > 1);
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    assert(stream_handle != NULL);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    if (use_feature && use_dimension) {
        less_tensor_kernel_feature_dimension<<<blocks, threads, 0, stream>>>(
            input_data, middle_data, thres_data, size);
    } else if (!use_feature && use_dimension) {
        less_tensor_kernel_dimension<<<blocks, threads, 0, stream>>>(
            input_data, middle_data, thres_data, size, dim);
    } else if (use_feature && !use_dimension) {
        less_tensor_kernel_feature<<<blocks, threads, 0, stream>>>(
            input_data, middle_data, thres_data, size, dim);
    } else {
        less_tensor_kernel_global<<<blocks, threads, 0, stream>>>(
            input_data, middle_data, thres_data, size);
    }
    return DLGpuReduceSum(middle, output, axes, num_ax, stream_handle);
}

__global__ void multiple_grouping_alpha_kernel(float *arr, const int *grouping,
                                               const float *alpha, size_t size,
                                               size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    int group_ind = grouping[ind / dim];
    float cur_alpha = alpha[group_ind * dim + ind % dim];
    arr[ind] *= cur_alpha;
}

int DLGpuMultiplyGroupingAlpha(DLArrayHandle arr, const DLArrayHandle grouping,
                               const DLArrayHandle alpha,
                               DLStreamHandle stream_handle = NULL) {
    assert(arr->ndim == 2);
    size_t size = ArrSize(arr);
    size_t dim = arr->shape[1];
    float *arr_data = (float *)arr->data;
    const int *grouping_data = (const int *)grouping->data;
    const float *alpha_data = (const float *)alpha->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    assert(stream_handle != NULL);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    multiple_grouping_alpha_kernel<<<blocks, threads, 0, stream>>>(
        arr_data, grouping_data, alpha_data, size, dim);
    return 0;
}