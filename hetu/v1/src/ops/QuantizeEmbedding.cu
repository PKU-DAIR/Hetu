#include "hip/hip_runtime.h"
#include "gpu_runtime.h"
#include "gpu_functions.cuh"
#include "random.h"

template <class T>
__global__ void prepack_kernel(const float *input, T *output, float *qparams,
                               HetuRandomState cudars, bool stochastic,
                               size_t dim, size_t rsize) {
    size_t rind = blockIdx.x * blockDim.x + threadIdx.x;
    if (rind >= rsize)
        return;
    size_t offset = dim * rind;
    const float *cur_input = input + offset;
    float *cur_qparam = qparams + 2 * rind;
    T *cur_out = output + offset;
    float maxele = cur_input[0];
    float minele = maxele;
    for (int i = 1; i < dim; ++i) {
        float cur_val = cur_input[i];
        maxele = max(maxele, cur_val);
        minele = min(minele, cur_val);
    }
    float range = maxele - minele;
    float scale = range / __numeric_limits_on_device<T>();
    cur_qparam[0] = scale;
    cur_qparam[1] = minele;
    for (int i = 0; i < dim; ++i) {
        float cur_value = cur_input[i];
        T out;
        if (stochastic) {
            out =
                stochastic_rounding<T>(cur_value, scale, minele, cudars, rind);
        } else {
            out = fixed_rounding<T>(cur_value, scale, minele);
        }
        cur_out[i] = out;
    }
}

template <class T>
__global__ void quantized_embedding_lookup_kernel(const T *input,
                                                  const int *indices,
                                                  float *output, float *qparams,
                                                  size_t dim, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    int cur_ind = indices[ind];
    const T *cur_input = input + dim * cur_ind;
    float *cur_qpar = qparams + 2 * cur_ind;
    float *cur_output = output + dim * ind;
    float scale = cur_qpar[0];
    float zero_point = cur_qpar[1];
    for (int i = 0; i < dim; ++i) {
        cur_output[i] = float(cur_input[i]) * scale + zero_point;
    }
}

int DLGpuPrepackEmbedding(const DLArrayHandle input, DLArrayHandle output,
                          DLArrayHandle qparams, int digit,
                          DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t rsize = input->shape[0];
    size_t dim = input->shape[1];
    const float *input_data = (const float *)input->data;
    float *qparam_data = (float *)qparams->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, rsize);
    HetuRandomState &cudars = GetRandomState(dim);
    if (digit == 8) {
        uint8_t *output_data = (uint8_t *)output->data;

        if (stream_handle)
            prepack_kernel<uint8_t><<<blocks, threads, 0,
                                      *(hipStream_t *)stream_handle->handle>>>(
                input_data, output_data, qparam_data, cudars, true, dim, rsize);
        else
            prepack_kernel<uint8_t><<<blocks, threads>>>(
                input_data, output_data, qparam_data, cudars, true, dim, rsize);
    } else if (digit == 16) {
        uint16_t *output_data = (uint16_t *)output->data;

        if (stream_handle)
            prepack_kernel<uint16_t><<<
                blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                input_data, output_data, qparam_data, cudars, true, dim, rsize);
        else
            prepack_kernel<uint16_t><<<blocks, threads>>>(
                input_data, output_data, qparam_data, cudars, true, dim, rsize);
    } else {
        assert(false);
    }
    return 0;
}

int DLGpuQuantizedEmbeddingLookup(const DLArrayHandle input,
                                  const DLArrayHandle indices,
                                  DLArrayHandle output, DLArrayHandle qparams,
                                  int digit,
                                  DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = ArrSize(indices);
    size_t dim = input->shape[1];
    const int *indices_data = (const int *)indices->data;
    float *qparam_data = (float *)qparams->data;
    float *output_data = (float *)output->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (digit == 8) {
        uint8_t *input_data = (uint8_t *)input->data;

        if (stream_handle)
            quantized_embedding_lookup_kernel<uint8_t><<<
                blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                input_data, indices_data, output_data, qparam_data, dim, size);
        else
            quantized_embedding_lookup_kernel<uint8_t><<<blocks, threads>>>(
                input_data, indices_data, output_data, qparam_data, dim, size);
    } else if (digit == 16) {
        uint16_t *input_data = (uint16_t *)input->data;

        if (stream_handle)
            quantized_embedding_lookup_kernel<uint16_t><<<
                blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                input_data, indices_data, output_data, qparam_data, dim, size);
        else
            quantized_embedding_lookup_kernel<uint16_t><<<blocks, threads>>>(
                input_data, indices_data, output_data, qparam_data, dim, size);

    } else {
        assert(false);
    }
    return 0;
}

template <class T>
__global__ void dequantize_lookup_kernel(const T *input, const int *indices,
                                         float *output, float scale,
                                         float minele, size_t nrow, size_t dim,
                                         size_t size) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return;

    int id = indices[index];
    float *output_ptr = output + dim * index;
    if (id < 0 || id >= nrow) {
        for (int i = 0; i < dim; i++)
            output_ptr[i] = 0;
    } else {
        const T *input_ptr = input + dim * id;
        for (int i = 0; i < dim; i++) {
            output_ptr[i] = (float)input_ptr[i] * scale + minele;
        }
    }
}

int DLGpuUnifiedQuantizedEmbeddingLookup(const DLArrayHandle input,
                                         const DLArrayHandle indices,
                                         DLArrayHandle output, int digit,
                                         float scale, float minele,
                                         DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = ArrSize(indices);
    size_t nrow = input->shape[0];
    size_t dim = input->shape[1];
    const int *indices_data = (const int *)indices->data;
    float *output_data = (float *)output->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (digit == 8) {
        uint8_t *input_data = (uint8_t *)input->data;

        if (stream_handle)
            dequantize_lookup_kernel<<<
                blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                input_data, indices_data, output_data, scale, minele, nrow, dim,
                size);
        else
            dequantize_lookup_kernel<<<blocks, threads>>>(
                input_data, indices_data, output_data, scale, minele, nrow, dim,
                size);
    } else if (digit == 16) {
        uint16_t *input_data = (uint16_t *)input->data;

        if (stream_handle)
            dequantize_lookup_kernel<<<
                blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
                input_data, indices_data, output_data, scale, minele, nrow, dim,
                size);
        else
            dequantize_lookup_kernel<<<blocks, threads>>>(
                input_data, indices_data, output_data, scale, minele, nrow, dim,
                size);

    } else {
        assert(false);
    }
    return 0;
}
