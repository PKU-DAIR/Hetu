#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

const float EPSILON = 1e-12;

// -label * log(prediction) - (1 - label) * log(1 - prediction)
__global__ void binary_cross_entropy_kernel(int nrow, const float *prediction,
                                            const float *label, float *loss) {
    // Two dimensional thread blocks.
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nrow)
        return;
    float cur_pred = prediction[id];
    float cur_label = label[id];
    assert(cur_label >= 0 && cur_label <= 1);
    float log_input_val = log(cur_pred + EPSILON);
    float log_1_minus_input_val = log(1 - cur_pred + EPSILON);
    // float log_input_val = log(cur_pred);
    // float log_1_minus_input_val = log(1 - cur_pred);

    // float neg_100 = -100;
    // log_input_val = max(log_input_val, neg_100);
    // log_1_minus_input_val = max(log_1_minus_input_val, neg_100);
    loss[id] =
        ((cur_label - 1) * log_1_minus_input_val) - (cur_label * log_input_val);
}

int DLGpuBinaryCrossEntropy(const DLArrayHandle prediction,
                            const DLArrayHandle label, DLArrayHandle loss,
                            DLStreamHandle stream_handle = NULL) {
    size_t indim = prediction->ndim;
    assert(indim == label->ndim && indim == loss->ndim);
    int nrow = 1;
    for (int i = 0; i < indim - 1; ++i) {
        nrow *= prediction->shape[i];
    }

    const float *prediction_data = (const float *)prediction->data;
    const float *label_data = (const float *)label->data;
    float *output_data = (float *)loss->data;

    dim3 blocks;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (nrow + 1023) / 1024;
    }
    // 1 block
    if (stream_handle) {
        binary_cross_entropy_kernel<<<blocks, threads, 0,
                                      *(hipStream_t *)stream_handle->handle>>>(
            nrow, prediction_data, label_data, output_data);
    } else {
        binary_cross_entropy_kernel<<<blocks, threads>>>(
            nrow, prediction_data, label_data, output_data);
    }
    return 0;
}

__global__ void binary_cross_entropy_gradient_kernel(int nrow,
                                                     const float *prediction,
                                                     const float *label,
                                                     const float *output_grad,
                                                     float *output) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nrow)
        return;
    float cur_pred = prediction[id];
    output[id] = output_grad[id] * (cur_pred - label[id])
                 / max((1 - cur_pred) * cur_pred, EPSILON);
}

int DLGpuBinaryCrossEntropy_Gradient(const DLArrayHandle prediction,
                                     const DLArrayHandle label,
                                     const DLArrayHandle output_grad,
                                     DLArrayHandle output,
                                     DLStreamHandle stream_handle = NULL) {
    size_t indim = prediction->ndim;
    assert(indim >= 2 && indim == label->ndim && indim == output_grad->ndim
           && indim == output->ndim);
    int nrow = 1;
    for (int i = 0; i < indim - 1; ++i) {
        nrow *= prediction->shape[i];
    }

    const float *prediction_data = (const float *)prediction->data;
    const float *label_data = (const float *)label->data;
    const float *output_grad_data = (const float *)output_grad->data;
    float *output_data = (float *)output->data;

    dim3 blocks;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (nrow + 1023) / 1024;
    }
    if (stream_handle) {
        binary_cross_entropy_gradient_kernel<<<
            blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
            nrow, prediction_data, label_data, output_grad_data, output_data);
    } else {
        binary_cross_entropy_gradient_kernel<<<blocks, threads>>>(
            nrow, prediction_data, label_data, output_grad_data, output_data);
    }
    return 0;
}
