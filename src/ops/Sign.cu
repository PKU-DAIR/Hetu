#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void sign_kernel(const float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    float result;
    if (input[ind] > 0)
        result = 1.0;
    else if (input[ind] < 0)
        result = -1.0;
    else
        result = 0.0;
    output[ind] = result;
}

int DLGpuSign(const DLArrayHandle input, DLArrayHandle output,
              DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(input);
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    const float *input_data = (const float *)input->data;
    float *output_data = (float *)output->data;
    if (stream_handle)
        sign_kernel<<<blocks, threads, 0,
                      *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        sign_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}
